#include "hip/hip_runtime.h"
#include "DenseMapper/denseMap.h"
#include <fstream>

__global__ void resetHashKernel(HashEntry *hashTable, int numEntry)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numEntry)
        return;

    hashTable[index].ptr = -1;
    hashTable[index].offset = -1;
}

__global__ void resetHeapKernel(int *heap, int *heapPtr, int numBlock)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numBlock)
        return;

    if (index == 0)
        heapPtr[0] = numBlock - 1;

    heap[index] = numBlock - index - 1;
}

void MapStruct::reset()
{
    dim3 block(1024);
    dim3 grid(div_up(hashTableSize, block.x));
    resetHashKernel<<<grid, block>>>(hashTable, hashTableSize);

    grid = dim3(div_up(voxelBlockSize, block.x));
    resetHeapKernel<<<grid, block>>>(heap, heapPtr, voxelBlockSize);

    hipMemset(excessPtr, 0, sizeof(int));
    hipMemset(bucketMutex, 0, sizeof(int) * bucketSize);
    hipMemset(voxelBlock, 0, sizeof(Voxel) * BlockSize3 * voxelBlockSize);
}

void MapStruct::create(
    int hashTableSize,
    int bucketSize,
    int voxelBlockSize,
    float voxelSize,
    float truncationDist)
{
    hipMalloc((void **)&excessPtr, sizeof(int));
    hipMalloc((void **)&heapPtr, sizeof(int));
    hipMalloc((void **)&visibleBlockNum, sizeof(uint));
    hipMalloc((void **)&bucketMutex, sizeof(int) * bucketSize);
    hipMalloc((void **)&heap, sizeof(int) * voxelBlockSize);
    hipMalloc((void **)&hashTable, sizeof(HashEntry) * hashTableSize);
    hipMalloc((void **)&visibleTable, sizeof(HashEntry) * hashTableSize);
    hipMalloc((void **)&voxelBlock, sizeof(Voxel) * voxelBlockSize * BlockSize3);

    this->hashTableSize = hashTableSize;
    this->bucketSize = bucketSize;
    this->voxelBlockSize = voxelBlockSize;
    this->voxelSize = voxelSize;
    this->truncationDist = truncationDist;
}

void MapStruct::release()
{
    hipFree((void *)heap);
    hipFree((void *)heapPtr);
    hipFree((void *)hashTable);
    hipFree((void *)bucketMutex);
    hipFree((void *)excessPtr);
    hipFree((void *)voxelBlock);
    hipFree((void *)visibleBlockNum);
    hipFree((void *)visibleTable);
}

void MapStruct::getVisibleBlockCount(uint &hostData)
{
    hipMemcpy(&hostData, visibleBlockNum, sizeof(uint), hipMemcpyDeviceToHost);
}

void MapStruct::resetVisibleBlockCount()
{
    hipMemset(visibleBlockNum, 0, sizeof(uint));
}

bool MapStruct::empty()
{
    return bucketSize == 0;
}