#include "hip/hip_runtime.h"
#include "localMapper/denseMap.h"
#include <fstream>

__global__ void resetHashKernel(HashEntry *hash_table, int numEntry)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numEntry)
        return;

    hash_table[index].ptr_ = -1;
    hash_table[index].offset_ = -1;
}

__global__ void resetHeapKernel(int *heap, int *heap_counter, int numBlock)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numBlock)
        return;

    heap[index] = numBlock - index - 1;

    if (index == 0)
    {
        heap_counter[0] = numBlock - 1;
    }
}

void MapStruct::reset()
{
    dim3 block(1024);
    dim3 grid(div_up(hashTableSize, block.x));
    resetHashKernel<<<grid, block>>>(hash_table_, hashTableSize);

    grid = dim3(div_up(voxelBlockSize, block.x));
    resetHeapKernel<<<grid, block>>>(heap_mem_, heap_mem_counter_, voxelBlockSize);

    hipMemset(excess_counter_, 0, sizeof(int));
    hipMemset(bucket_mutex_, 0, sizeof(int) * bucketSize);
    hipMemset(voxels_, 0, sizeof(Voxel) * voxelBlockSize);
}

void MapStruct::create(
    int hashTableSize,
    int bucketSize,
    int voxelBlockSize,
    float voxelSize,
    float truncationDist)
{
    hipMalloc((void **)&excess_counter_, sizeof(int));
    hipMalloc((void **)&heap_mem_counter_, sizeof(int));
    hipMalloc((void **)&visibleBlockNum, sizeof(uint));
    hipMalloc((void **)&bucket_mutex_, sizeof(int) * bucketSize);
    hipMalloc((void **)&heap_mem_, sizeof(int) * voxelBlockSize);
    hipMalloc((void **)&hash_table_, sizeof(HashEntry) * hashTableSize);
    hipMalloc((void **)&visibleTable, sizeof(HashEntry) * hashTableSize);
    hipMalloc((void **)&voxels_, sizeof(Voxel) * voxelBlockSize * BLOCK_SIZE3);

    this->hashTableSize = hashTableSize;
    this->bucketSize = bucketSize;
    this->voxelBlockSize = voxelBlockSize;
    this->voxelSize = voxelSize;
    this->truncationDist = truncationDist;
}

void MapStruct::release()
{
    hipFree((void *)heap_mem_);
    hipFree((void *)heap_mem_counter_);
    hipFree((void *)hash_table_);
    hipFree((void *)bucket_mutex_);
    hipFree((void *)excess_counter_);
    hipFree((void *)voxels_);
    hipFree((void *)visibleBlockNum);
    hipFree((void *)visibleTable);
}

bool MapStruct::empty()
{
    return bucketSize == 0;
}