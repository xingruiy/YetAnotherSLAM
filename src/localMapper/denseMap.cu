#include "hip/hip_runtime.h"
#include "localMapper/denseMap.h"
#include "utils/cudaUtils.h"

MapStruct::MapStruct()
    : numEntry(0), numBucket(0),
      numBlock(0), voxelSize(0),
      truncDist(0)
{
}

void MapStruct::create(int numEntry, int numBucket, int numBlock,
                       float voxelSize, float truncationDist)
{
    hipMalloc((void **)&excessPtr, sizeof(int));
    hipMalloc((void **)&heapPtr, sizeof(int));
    hipMalloc((void **)&numVisibleEntry, sizeof(uint));
    hipMalloc((void **)&bucketMutex, sizeof(int) * numBucket);
    hipMalloc((void **)&heap, sizeof(int) * numBlock);
    hipMalloc((void **)&hashTable, sizeof(HashEntry) * numEntry);
    hipMalloc((void **)&visibleEntry, sizeof(HashEntry) * numEntry);
    hipMalloc((void **)&voxelBlocks, sizeof(Voxel) * numBlock * BlockSize3);

    this->numEntry = numEntry;
    this->numBlock = numBlock;
    this->numBucket = numBucket;
    this->voxelSize = voxelSize;
    this->truncDist = truncationDist;

    reset();
}

void MapStruct::release()
{
    hipFree((void *)heap);
    hipFree((void *)heapPtr);
    hipFree((void *)hashTable);
    hipFree((void *)bucketMutex);
    hipFree((void *)excessPtr);
    hipFree((void *)voxelBlocks);
    hipFree((void *)numVisibleEntry);
    hipFree((void *)visibleEntry);

    this->numEntry = 0;
    this->numBlock = 0;
    this->numBucket = 0;
    this->voxelSize = 0;
    this->truncDist = 0;
}

bool MapStruct::empty()
{
    return numBlock == 0;
}

void MapStruct::resetNumVisibleEntry()
{
    hipMemset(numVisibleEntry, 0, sizeof(uint));
}

void MapStruct::getNumVisibleEntry(uint &hostData)
{
    hipMemcpy(&hostData, numVisibleEntry, sizeof(uint), hipMemcpyDeviceToHost);
}

__global__ void resetHashEntryKernel(HashEntry *hashTable, int numEntry)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= numEntry)
        return;

    hashTable[idx].offset = -1;
    hashTable[idx].ptr = -1;
}

__global__ void resetHeap(int *heap, int *heapPtr, int numBlock)
{
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= numBlock)
        return;

    if (idx == 0)
        heapPtr[0] = numBlock - 1;

    heap[idx] = numBlock - idx - 1;
}

void MapStruct::reset()
{
    dim3 block(1024);
    dim3 grid = getGridConfiguration1D(block, numEntry);

    resetHashEntryKernel<<<grid, block>>>(hashTable, numEntry);

    grid = getGridConfiguration1D(block, numBlock);
    resetHeap<<<grid, block>>>(heap, heapPtr, numBlock);

    hipMemset((void *)excessPtr, 0, sizeof(int));
    hipMemset((void *)bucketMutex, 0, sizeof(int) * numBucket);
    hipMemset((void *)voxelBlocks, 0, sizeof(Voxel) * numBlock * BlockSize);
}