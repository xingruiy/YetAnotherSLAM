#include "localMapper/denseMap.h"

MapStruct::MapStruct()
    : numEntry(0), numBucket(0),
      numBlock(0), voxelSize(0),
      truncDist(0)
{
}

void MapStruct::reset()
{
}

void MapStruct::create(int numEntry, int numBucket, int numBlock,
                       float voxelSize, float truncationDist)
{
    hipMalloc((void **)&excessPtr, sizeof(int));
    hipMalloc((void **)&heapPtr, sizeof(int));
    hipMalloc((void **)&numVisibleEntry, sizeof(uint));
    hipMalloc((void **)&bucketMutex, sizeof(int) * numBucket);
    hipMalloc((void **)&heap, sizeof(int) * numBlock);
    hipMalloc((void **)&hashTable, sizeof(HashEntry) * numEntry);
    hipMalloc((void **)&visibleEntry, sizeof(HashEntry) * numEntry);
    hipMalloc((void **)&voxelBlocks, sizeof(Voxel) * numBlock * BlockSize3);

    this->numEntry = numEntry;
    this->numBlock = numBlock;
    this->numBucket = numBucket;
    this->voxelSize = voxelSize;
    this->truncDist = truncationDist;
}

void MapStruct::release()
{
    hipFree((void *)heap);
    hipFree((void *)heapPtr);
    hipFree((void *)hashTable);
    hipFree((void *)bucketMutex);
    hipFree((void *)excessPtr);
    hipFree((void *)voxelBlocks);
    hipFree((void *)numVisibleEntry);
    hipFree((void *)visibleEntry);

    this->numEntry = 0;
    this->numBlock = 0;
    this->numBucket = 0;
    this->voxelSize = 0;
    this->truncDist = 0;
}

bool MapStruct::empty()
{
    return numBlock == 0;
}

void MapStruct::resetNumVisibleEntry()
{
    hipMemset(numVisibleEntry, 0, sizeof(uint));
}

void MapStruct::getNumVisibleEntry(uint &hostData)
{
    hipMemcpy(&hostData, numVisibleEntry, sizeof(uint), hipMemcpyDeviceToHost);
}