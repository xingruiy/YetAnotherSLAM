#include "hip/hip_runtime.h"
#include "localMapper/denseMap.h"
#include <fstream>

__global__ void resetHashKernel(HashEntry *hashTable, int numEntry)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numEntry)
        return;

    hashTable[index].ptr = -1;
    hashTable[index].offset = -1;
}

__global__ void resetHeapKernel(int *heap, int *heapPtr, int numBlock)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numBlock)
        return;

    if (index == 0)
        heapPtr[0] = numBlock - 1;

    heap[index] = numBlock - index - 1;
}

void MapStruct::reset()
{
    dim3 block(1024);
    dim3 grid(div_up(hashTableSize, block.x));
    resetHashKernel<<<grid, block>>>(hash_table_, hashTableSize);

    grid = dim3(div_up(voxelBlockSize, block.x));
    resetHeapKernel<<<grid, block>>>(heap_mem_, heap_mem_counter_, voxelBlockSize);

    hipMemset(excess_counter_, 0, sizeof(int));
    hipMemset(bucket_mutex_, 0, sizeof(int) * bucketSize);
    hipMemset(voxels_, 0, sizeof(Voxel) * BlockSize3 * voxelBlockSize);
}

void MapStruct::create(
    int hashTableSize,
    int bucketSize,
    int voxelBlockSize,
    float voxelSize,
    float truncationDist)
{
    hipMalloc((void **)&excess_counter_, sizeof(int));
    hipMalloc((void **)&heap_mem_counter_, sizeof(int));
    hipMalloc((void **)&visibleBlockNum, sizeof(uint));
    hipMalloc((void **)&bucket_mutex_, sizeof(int) * bucketSize);
    hipMalloc((void **)&heap_mem_, sizeof(int) * voxelBlockSize);
    hipMalloc((void **)&hash_table_, sizeof(HashEntry) * hashTableSize);
    hipMalloc((void **)&visibleTable, sizeof(HashEntry) * hashTableSize);
    hipMalloc((void **)&voxels_, sizeof(Voxel) * voxelBlockSize * BlockSize3);

    this->hashTableSize = hashTableSize;
    this->bucketSize = bucketSize;
    this->voxelBlockSize = voxelBlockSize;
    this->voxelSize = voxelSize;
    this->truncationDist = truncationDist;
}

void MapStruct::release()
{
    hipFree((void *)heap_mem_);
    hipFree((void *)heap_mem_counter_);
    hipFree((void *)hash_table_);
    hipFree((void *)bucket_mutex_);
    hipFree((void *)excess_counter_);
    hipFree((void *)voxels_);
    hipFree((void *)visibleBlockNum);
    hipFree((void *)visibleTable);
}

void MapStruct::getVisibleBlockCount(uint &hostData)
{
    hipMemcpy(&hostData, visibleBlockNum, sizeof(uint), hipMemcpyDeviceToHost);
}

void MapStruct::resetVisibleBlockCount()
{
    hipMemset(visibleBlockNum, 0, sizeof(uint));
}

bool MapStruct::empty()
{
    return bucketSize == 0;
}