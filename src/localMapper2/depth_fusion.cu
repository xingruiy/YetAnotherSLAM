#include "hip/hip_runtime.h"
#include "map_proc.h"
#include <opencv2/opencv.hpp>
#include <opencv2/cudaarithm.hpp>
#include <thrust/device_vector.h>
#include "utils/numType.h"
#include "utils/cudaUtils.h"

__device__ inline bool is_vertex_visible(
    Vec3f pt, SE3f inv_pose,
    int cols, int rows, float fx,
    float fy, float cx, float cy,
    float depthMin, float depthMax)
{
    pt = inv_pose * (pt);
    Vec2f pt2d = Vec2f(fx * pt(0) / pt(2) + cx, fy * pt(1) / pt(2) + cy);
    return !(pt2d(0) < 0 || pt2d(1) < 0 ||
             pt2d(0) > cols - 1 || pt2d(1) > rows - 1 ||
             pt(2) < depthMin || pt(2) > depthMax);
}

__device__ inline bool is_block_visible(
    const Vec3i &block_pos,
    SE3f inv_pose, const float &voxelSize,
    int cols, int rows, float fx,
    float fy, float cx, float cy,
    float depthMin, float depthMax)
{
    float scale = voxelSize * BLOCK_SIZE; //param.block_size_metric();
#pragma unroll
    for (int corner = 0; corner < 8; ++corner)
    {
        Vec3i tmp = block_pos;
        tmp(0) += (corner & 1) ? 1 : 0;
        tmp(1) += (corner & 2) ? 1 : 0;
        tmp(2) += (corner & 4) ? 1 : 0;

        if (is_vertex_visible(tmp.cast<float>() * scale, inv_pose, cols, rows, fx, fy, cx, cy, depthMin, depthMax))
            return true;
    }

    return false;
}

__global__ void check_visibility_flag_kernel(
    MapStruct map_struct, uchar *flag, SE3f inv_pose,
    int cols, int rows, float fx, float fy, float cx, float cy, float voxelSize,
    float depthMin, float depthMax)
{
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= map_struct.hashTableSize)
        return;

    HashEntry &current = map_struct.hash_table_[idx];
    if (current.ptr_ != -1)
    {
        switch (flag[idx])
        {
        default:
        {
            if (is_block_visible(current.pos_, inv_pose, voxelSize, cols, rows, fx, fy, cx, cy, depthMin, depthMax))
            {
                flag[idx] = 1;
            }
            else
            {
                current.ptr_ = -1;
                flag[idx] = 0;
            }

            return;
        }
        case 2:
        {
            flag[idx] = 1;
            return;
        }
        }
    }
}

__global__ void copy_visible_block_kernel(HashEntry *hash_table, HashEntry *visible_block, int hashTableSize, const uchar *flag, const int *pos)
{
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx >= hashTableSize)
        return;

    if (flag[idx] == 1)
        visible_block[pos[idx]] = hash_table[idx];
}

__device__ inline Vec2f project(
    Vec3f pt, float fx, float fy, float cx, float cy)
{
    return Vec2f(fx * pt(0) / pt(2) + cx, fy * pt(1) / pt(2) + cy);
}

__device__ inline Vec3f unproject(
    int x, int y, float z, float invfx, float invfy, float cx, float cy)
{
    return Vec3f(invfx * (x - cx) * z, invfy * (y - cy) * z, z);
}

__device__ inline Vec3f unproject_world(
    int x, int y, float z, float invfx,
    float invfy, float cx, float cy, SE3f pose)
{
    return pose * (unproject(x, y, z, invfx, invfy, cx, cy));
}

__global__ void create_blocks_kernel(MapStruct map_struct, cv::cuda::PtrStepSz<float> depth,
                                     float invfx, float invfy, float cx, float cy,
                                     SE3f pose, uchar *flag, float depthMin, float depthMax)
{
    const int x = threadIdx.x + blockDim.x * blockIdx.x;
    const int y = threadIdx.y + blockDim.y * blockIdx.y;
    if (x >= depth.cols || y >= depth.rows)
        return;

    float z = depth.ptr(y)[x];
    if (isnan(z) || z < depthMin || z > depthMax)
        return;

    float z_thresh = map_struct.truncationDist * 0.5;
    float z_near = max(depthMin, z - z_thresh);
    float z_far = min(depthMax, z + z_thresh);
    if (z_near >= z_far)
        return;

    Vec3i block_near = voxelPosToBlockPos(worldPtToVoxelPos(unproject_world(x, y, z_near, invfx, invfy, cx, cy, pose), map_struct.voxelSize));
    Vec3i block_far = voxelPosToBlockPos(worldPtToVoxelPos(unproject_world(x, y, z_far, invfx, invfy, cx, cy, pose), map_struct.voxelSize));

    Vec3i d = block_far - block_near;
    Vec3i increment = Vec3i(d(0) < 0 ? -1 : 1, d(1) < 0 ? -1 : 1, d(2) < 0 ? -1 : 1);
    Vec3i incre_abs = Vec3i(abs(d(0)), abs(d(1)), abs(d(2)));
    Vec3i incre_err = Vec3i(incre_abs(0) << 1, incre_abs(1) << 1, incre_abs(2) << 1);

    int err_1;
    int err_2;

    // Bresenham's line algorithm
    // details see : https://en.m.wikipedia.org/wiki/Bresenham%27s_line_algorithm
    if ((incre_abs(0) >= incre_abs(1)) && (incre_abs(0) >= incre_abs(2)))
    {
        err_1 = incre_err(1) - 1;
        err_2 = incre_err(2) - 1;
        createBlock(block_near,
                    map_struct.heap_mem_,
                    map_struct.heap_mem_counter_,
                    map_struct.hash_table_,
                    map_struct.bucket_mutex_,
                    map_struct.excess_counter_,
                    map_struct.hashTableSize,
                    map_struct.bucketSize);
        for (int i = 0; i < incre_abs(0); ++i)
        {
            if (err_1 > 0)
            {
                block_near(1) += increment(1);
                err_1 -= incre_err(0);
            }

            if (err_2 > 0)
            {
                block_near(2) += increment(2);
                err_2 -= incre_err(0);
            }

            err_1 += incre_err(1);
            err_2 += incre_err(2);
            block_near(0) += increment(0);
            createBlock(block_near,
                        map_struct.heap_mem_,
                        map_struct.heap_mem_counter_,
                        map_struct.hash_table_,
                        map_struct.bucket_mutex_,
                        map_struct.excess_counter_,
                        map_struct.hashTableSize,
                        map_struct.bucketSize);
        }
    }
    else if ((incre_abs(1) >= incre_abs(0)) && (incre_abs(1) >= incre_abs(2)))
    {
        err_1 = incre_err(0) - 1;
        err_2 = incre_err(2) - 1;
        createBlock(block_near,
                    map_struct.heap_mem_,
                    map_struct.heap_mem_counter_,
                    map_struct.hash_table_,
                    map_struct.bucket_mutex_,
                    map_struct.excess_counter_,
                    map_struct.hashTableSize,
                    map_struct.bucketSize);
        for (int i = 0; i < incre_abs(1); ++i)
        {
            if (err_1 > 0)
            {
                block_near(0) += increment(0);
                err_1 -= incre_err(1);
            }

            if (err_2 > 0)
            {
                block_near(2) += increment(2);
                err_2 -= incre_err(1);
            }

            err_1 += incre_err(0);
            err_2 += incre_err(2);
            block_near(1) += increment(1);
            createBlock(block_near,
                        map_struct.heap_mem_,
                        map_struct.heap_mem_counter_,
                        map_struct.hash_table_,
                        map_struct.bucket_mutex_,
                        map_struct.excess_counter_,
                        map_struct.hashTableSize,
                        map_struct.bucketSize);
        }
    }
    else
    {
        err_1 = incre_err(1) - 1;
        err_2 = incre_err(0) - 1;
        createBlock(block_near,
                    map_struct.heap_mem_,
                    map_struct.heap_mem_counter_,
                    map_struct.hash_table_,
                    map_struct.bucket_mutex_,
                    map_struct.excess_counter_,
                    map_struct.hashTableSize,
                    map_struct.bucketSize);
        for (int i = 0; i < incre_abs(2); ++i)
        {
            if (err_1 > 0)
            {
                block_near(1) += increment(1);
                err_1 -= incre_err(2);
            }

            if (err_2 > 0)
            {
                block_near(0) += increment(0);
                err_2 -= incre_err(2);
            }

            err_1 += incre_err(1);
            err_2 += incre_err(0);
            block_near(2) += increment(2);
            createBlock(block_near,
                        map_struct.heap_mem_,
                        map_struct.heap_mem_counter_,
                        map_struct.hash_table_,
                        map_struct.bucket_mutex_,
                        map_struct.excess_counter_,
                        map_struct.hashTableSize,
                        map_struct.bucketSize);
        }
    }
}

__global__ void update_map_kernel(MapStruct map_struct,
                                  HashEntry *visible_blocks,
                                  uint count_visible_block,
                                  cv::cuda::PtrStepSz<float> depth,
                                  SE3f inv_pose,
                                  float fx, float fy,
                                  float cx, float cy,
                                  float depthMin, float depthMax)
{
    if (blockIdx.x >= map_struct.hashTableSize || blockIdx.x >= count_visible_block)
        return;

    HashEntry &current = visible_blocks[blockIdx.x];

    Vec3i voxel_pos = blockPosToVoxelPos(current.pos_);
    float dist_thresh = map_struct.truncationDist;
    float inv_dist_thresh = 1.0 / dist_thresh;

#pragma unroll
    for (int block_idx_z = 0; block_idx_z < 8; ++block_idx_z)
    {
        Vec3i local_pos = Vec3i(threadIdx.x, threadIdx.y, block_idx_z);
        Vec3f pt = inv_pose * (voxelPosToWorldPt(voxel_pos + local_pos, map_struct.voxelSize));

        int u = __float2int_rd(fx * pt(0) / pt(2) + cx + 0.5);
        int v = __float2int_rd(fy * pt(1) / pt(2) + cy + 0.5);
        if (u < 0 || v < 0 || u > depth.cols - 1 || v > depth.rows - 1)
            continue;

        float dist = depth.ptr(v)[u];
        if (isnan(dist) || dist < 1e-2 || dist > depthMax || dist < depthMin)
            continue;

        float sdf = dist - pt(2);
        if (sdf < -dist_thresh)
            continue;

        sdf = fmin(1.0f, sdf * inv_dist_thresh);
        const int local_idx = localPosToLocalIdx(local_pos);
        Voxel &voxel = map_struct.voxels_[current.ptr_ + local_idx];

        auto sdf_p = unpackFloat(voxel.sdf);
        auto weight_p = voxel.weight;
        // auto weight = 1 / (dist);

        if (weight_p == 0)
        {
            voxel.sdf = packFloat(sdf);
            voxel.weight = 1;
            continue;
        }

        // fuse depth
        sdf_p = (sdf_p * weight_p + sdf) / (weight_p + 1);
        voxel.sdf = packFloat(sdf_p);
        voxel.weight = (weight_p + 1);
    }
}

// __global__ void update_map_with_colour_kernel(MapStorage map_struct,
//                                               HashEntry *visible_blocks,
//                                               uint count_visible_block,
//                                               cv::cuda::PtrStepSz<float> depth,
//                                               cv::cuda::PtrStepSz<Vec3c> image,
//                                               SE3f inv_pose,
//                                               float fx, float fy,
//                                               float cx, float cy)
// {
//     if (blockIdx.x >= param.num_total_hash_entries_ || blockIdx.x >= count_visible_block)
//         return;

//     HashEntry &current = visible_blocks[blockIdx.x];

//     Vec3i voxel_pos = blockPosToVoxelPos(current.pos_);
//     float dist_thresh = param.truncation_dist();
//     float inv_dist_thresh = 1.0 / dist_thresh;

// #pragma unroll
//     for (int block_idx_z = 0; block_idx_z < 8; ++block_idx_z)
//     {
//         Vec3i local_pos = Vec3i(threadIdx.x, threadIdx.y, block_idx_z);
//         Vec3f pt = inv_pose(voxelPosToWorldPt(voxel_pos + local_pos, map_struct.voxelSize));

//         int u = __float2int_rd(fx * pt.x / pt.z + cx + 0.5);
//         int v = __float2int_rd(fy * pt.y / pt.z + cy + 0.5);
//         if (u < 0 || v < 0 || u > depth.cols - 1 || v > depth.rows - 1)
//             continue;

//         float dist = depth.ptr(v)[u];
//         if (isnan(dist) || dist < 1e-2 || dist > depthMax || dist < depthMin)
//             continue;

//         float sdf = dist - pt.z;
//         if (sdf < -dist_thresh)
//             continue;

//         sdf = fmin(1.0f, sdf * inv_dist_thresh);
//         const int local_idx = localPosToLocalIdx(local_pos);
//         Voxel &voxel = map_struct.voxels_[current.ptr_ + local_idx];

//         auto sdf_p = voxel.getSDF();
//         auto weight_p = voxel.getWeight();
//         auto weight = 1 / (dist * dist);

//         // update colour
//         auto colour_new = image.ptr(v)[u];
//         auto colour_p = voxel.rgb;

//         if (voxel.weight == 0)
//         {
//             voxel.setSDF(sdf);
//             voxel.setWeight(weight);
//             voxel.rgb = colour_new;
//             continue;
//         }

//         // fuse depth
//         sdf_p = (sdf_p * weight_p + sdf * weight) / (weight_p + weight);
//         voxel.setSDF(sdf_p);
//         voxel.setWeight(weight_p + weight);

//         // fuse colour
//         colour_p = ToVec3c((colour_p * weight_p + colour_new * weight) / (weight_p + weight));
//         voxel.rgb = colour_p;
//     }
// }

// __global__ void update_map_weighted_kernel(
//     MapStorage map_struct,
//     HashEntry *visible_blocks,
//     uint count_visible_block,
//     cv::cuda::PtrStepSz<float> depth,
//     cv::cuda::PtrStepSz<Vec4f> normal,
//     cv::cuda::PtrStepSz<Vec3c> image,
//     SE3f inv_pose,
//     float fx, float fy,
//     float cx, float cy)
// {
//     if (blockIdx.x >= param.num_total_hash_entries_ || blockIdx.x >= count_visible_block)
//         return;

//     HashEntry &current = visible_blocks[blockIdx.x];

//     if (current.ptr_ < 0)
//         return;

//     Vec3i voxel_pos = blockPosToVoxelPos(current.pos_);
//     float dist_thresh = param.truncation_dist();
//     float inv_dist_thresh = 1.0 / dist_thresh;

// #pragma unroll
//     for (int block_idx_z = 0; block_idx_z < 8; ++block_idx_z)
//     {
//         Vec3i local_pos = Vec3i(threadIdx.x, threadIdx.y, block_idx_z);
//         Vec3f pt = inv_pose(voxelPosToWorldPt(voxel_pos + local_pos, map_struct.voxelSize));

//         int u = __float2int_rd(fx * pt(0) / pt.z + cx + 0.5);
//         int v = __float2int_rd(fy * pt.y / pt.z + cy + 0.5);
//         if (u < 0 || v < 0 || u > depth.cols - 1 || v > depth.rows - 1)
//             continue;

//         float dist = depth.ptr(v)[u];
//         auto n_c = ToVec3(normal.ptr(v)[u]);
//         if (isnan(dist) || isnan(n_c(0)) || dist > depthMax || dist < depthMin)
//             continue;

//         float sdf = dist - pt.z;
//         if (sdf < -dist_thresh)
//             continue;

//         sdf = fmin(1.0f, sdf * inv_dist_thresh);
//         const int local_idx = localPosToLocalIdx(local_pos);
//         Voxel &voxel = map_struct.voxels_[current.ptr_ + local_idx];

//         auto sdf_p = voxel.getSDF();
//         auto weight_p = voxel.getWeight();
//         auto weight = abs(sin(n_c.z)) / (dist * dist);

//         // update colour
//         auto colour_new = image.ptr(v)[u];
//         auto colour_p = voxel.rgb;

//         if (voxel.weight == 0)
//         {
//             voxel.setSDF(sdf);
//             voxel.setWeight(weight);
//             voxel.rgb = colour_new;
//             continue;
//         }

//         // fuse depth
//         sdf_p = (sdf_p * weight_p + sdf * weight) / (weight_p + weight);
//         voxel.setSDF(sdf_p);
//         voxel.setWeight(weight_p + weight);

//         // fuse colour
//         colour_p = ToVec3c((colour_p * weight_p + colour_new * weight) / (weight_p + weight));
//         voxel.rgb = colour_p;
//     }
// }

void update(
    MapStruct map_struct,
    // MapState state,
    const cv::cuda::GpuMat depth,
    const cv::cuda::GpuMat image,
    const Sophus::SE3d &frame_pose,
    const Eigen::Matrix3d &K,
    cv::cuda::GpuMat &cv_flag,
    cv::cuda::GpuMat &cv_pos_array,
    HashEntry *visible_blocks,
    uint &visible_block_count)
{
    if (cv_flag.empty())
        cv_flag.create(1, map_struct.hashTableSize, CV_8UC1);
    if (cv_pos_array.empty())
        cv_pos_array.create(1, map_struct.hashTableSize, CV_32SC1);

    thrust::device_ptr<uchar> flag(cv_flag.ptr<uchar>());
    thrust::device_ptr<int> pos_array(cv_pos_array.ptr<int>());

    float fx = K(0, 0);
    float fy = K(1, 1);
    float cx = K(0, 2);
    float cy = K(1, 2);
    float invfx = 1.0 / fx;
    float invfy = 1.0 / fy;

    const int cols = depth.cols;
    const int rows = depth.rows;

    dim3 thread(8, 8);
    dim3 block(div_up(cols, thread.x), div_up(rows, thread.y));

    create_blocks_kernel<<<block, thread>>>(
        map_struct,
        depth,
        invfx,
        invfy,
        cx, cy,
        frame_pose.cast<float>(),
        flag.get(),
        0.1f,
        3.0f);

    thread = dim3(1024);
    block = dim3(div_up(map_struct.hashTableSize, thread.x));

    check_visibility_flag_kernel<<<block, thread>>>(
        map_struct,
        flag.get(),
        frame_pose.inverse().cast<float>(),
        cols, rows,
        fx, fy,
        cx, cy,
        map_struct.voxelSize,
        0.1f,
        3.0f);

    thrust::exclusive_scan(flag, flag + map_struct.hashTableSize, pos_array);

    copy_visible_block_kernel<<<block, thread>>>(
        map_struct.hash_table_,
        visible_blocks,
        map_struct.hashTableSize,
        flag.get(),
        pos_array.get());

    visible_block_count = pos_array[map_struct.hashTableSize - 1];

    if (visible_block_count == 0)
        return;

    thread = dim3(8, 8);
    block = dim3(visible_block_count);

    // update_map_with_colour_kernel<<<block, thread>>>(
    //     map_struct,
    //     visible_blocks,
    //     visible_block_count,
    //     depth, image,
    //     frame_pose.inverse().cast<float>().matrix3x4(),
    //     fx, fy,
    //     cx, cy);
    update_map_kernel<<<block, thread>>>(
        map_struct,
        visible_blocks,
        visible_block_count,
        depth,
        frame_pose.inverse().cast<float>(),
        fx, fy,
        cx, cy,
        0.1f,
        3.0f);
}
