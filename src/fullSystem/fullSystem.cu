#include "hip/hip_runtime.h"
#include "fullSystem/fullSystem.h"
#include "localizer/localizer.h"
#include "denseTracker/cudaImageProc.h"

FullSystem::FullSystem(
    int w, int h,
    Mat33d K,
    int numLvl,
    bool enableViewer)
    : state(SystemState::NotInitialized),
      lastState(SystemState::NotInitialized),
      viewerEnabled(enableViewer),
      mappingEnabled(true),
      imageWidth(w),
      imageHeight(h),
      camIntrinsics(K),
      numProcessedFrames(0),
      useGraphMatching(false),
      shouldCalculateNormal(false)
{
    map = std::make_shared<Map>();
    localOptimizer = std::make_shared<LocalOptimizer>(K, 3, map);
    localMapper = std::make_shared<DenseMapping>(w, h, K);
    coarseTracker = std::make_shared<DenseTracker>(w, h, K, numLvl);

    lastTrackedPose = SE3(Mat44d::Identity());
    accumulateTransform = SE3(Mat44d::Identity());

    gpuBufferVec4FloatWxH.create(h, w, CV_32FC4);
    gpuBufferFloatWxH.create(h, w, CV_32FC1);

    localOptThread = std::thread(&LocalOptimizer::loop, localOptimizer.get());
}

FullSystem::~FullSystem()
{
    localOptimizer->setShouldQuit();
    printf("wating other threads to finish...\n");
    localOptThread.join();
    printf("all threads finished!\n");
}

void FullSystem::setCurrentNormal(GMat nmap)
{
    nmap.download(cpuBufferVec4FloatWxH);
}

void FullSystem::processFrame(Mat rawImage, Mat rawDepth)
{
    rawImage.convertTo(cpuBufferVec3FloatWxH, CV_32FC3);
    cv::cvtColor(cpuBufferVec3FloatWxH, cpuBufferFloatWxH, cv::COLOR_RGB2GRAY);

    currentFrame = std::make_shared<Frame>(
        imageWidth,
        imageHeight,
        camIntrinsics,
        rawImage,
        rawDepth,
        cpuBufferFloatWxH);

    switch (state)
    {
    case SystemState::NotInitialized:
    {
        if (viewerEnabled && viewer)
            viewer->setCurrentState(-1);

        coarseTracker->setReferenceFrame(currentFrame);
        createNewKF();
        fuseCurrentFrame();
        state = SystemState::OK;

        if (viewerEnabled && viewer)
            viewer->setCurrentState(0);

        break;
    }
    case SystemState::OK:
    {
        auto rval = trackCurrentFrame();
        if (rval)
        {
            fuseCurrentFrame();
            raytraceCurrentFrame();

            if (needNewKF())
            {
                createNewKF();
            }
            else
            {
                map->addFramePose(currentFrame->getTrackingResult(), currentKeyframe);
            }

            if (viewerEnabled && viewer)
                viewer->addTrackingResult(currentFrame->getPoseInLocalMap());
        }
        else
        {
            if (viewerEnabled && viewer)
                viewer->setCurrentState(1);

            state = SystemState::Lost;
        }

        break;
    }
    case SystemState::Lost:
    {
        printf("tracking loast, attempt to resuming...\n");
        if (tryRelocalizeCurrentFrame())
        {
            if (viewerEnabled && viewer)
                viewer->setCurrentState(0);
        }
        break;
    }
    }

    lastState = state;
    if (state == SystemState::OK)
        numProcessedFrames++;
}

bool FullSystem::trackCurrentFrame()
{
    coarseTracker->setTrackingFrame(currentFrame);
    SE3 tRes = coarseTracker->getIncrementalTransform();
    // accumulated local transform
    accumulateTransform = accumulateTransform * tRes.inverse();
    currentFrame->setTrackingResult(accumulateTransform);
    currentFrame->setReferenceKF(currentKeyframe);

    return true;
}

void FullSystem::fuseCurrentFrame()
{

    auto currDepth = coarseTracker->getReferenceDepth();
    localMapper->fuseFrame(currDepth, currentFrame->getPoseInLocalMap());
}

void FullSystem::raytraceCurrentFrame()
{
    localMapper->raytrace(gpuBufferVec4FloatWxH, currentFrame->getPoseInLocalMap());
    coarseTracker->setReferenceInvDepth(gpuBufferVec4FloatWxH);
    computeNormal(gpuBufferVec4FloatWxH, gpuBufferVec4FloatWxH2);
    currentFrame->setNormalMap(Mat(gpuBufferVec4FloatWxH2));
}

bool FullSystem::tryRelocalizeCurrentFrame()
{
    Mat descriptor;
    std::vector<bool> valid;
    std::vector<float> keyPointDepth;
    std::vector<Vec3f> keyPointNormal;
    std::vector<cv::KeyPoint> cvKeyPoint;
    auto matcher = std::make_shared<FeatureMatcher>(PointType::ORB, DescType::ORB);
    matcher->detectAndCompute(currentFrame->getImage(), cvKeyPoint, descriptor);
    matcher->computePointDepth(currentFrame->getDepth(), cvKeyPoint, keyPointDepth);

    if (shouldCalculateNormal)
        matcher->computePointNormal(cpuBufferVec4FloatWxH, cvKeyPoint, keyPointNormal);

    auto numFeatures = keyPointDepth.size();
    std::vector<Vec3d> keyPoint(numFeatures);

    if (numFeatures < 10)
    {
        printf("too few points detected(%lu), relocalization failed...\n", numFeatures);
        return false;
    }

    valid.resize(numFeatures);
    std::fill(valid.begin(), valid.end(), true);
    for (int n = 0; n < numFeatures; ++n)
    {
        auto &z = keyPointDepth[n];
        if (z > FLT_EPSILON)
        {
            auto &kp = cvKeyPoint[n].pt;
            keyPoint[n] = camIntrinsics.inverse() * Vec3d(kp.x, kp.y, 1.0) * z;
        }
        else
        {
            valid[n] = false;
        }
    }

    Localizer relocalizer;
    std::vector<SE3> hypothesesList;
    if (!relocalizer.getRelocHypotheses(
            map,
            keyPoint,
            keyPointNormal,
            descriptor,
            valid,
            hypothesesList,
            useGraphMatching))
        return false;

    if (hypothesesList.size() == 0)
    {
        printf("too few hypotheses(%lu), relocalization failed...\n", hypothesesList.size());
        return false;
    }

    if (viewerEnabled && viewer)
        viewer->setRelocalizationHypotheses(hypothesesList);

    return true;
}

bool FullSystem::needNewKF()
{
    auto dt = currentFrame->getTrackingResult();
    Vec3d t = dt.translation();
    if (t.norm() >= 0.3)
        return true;

    Vec3d r = dt.log().tail<3>();
    if (r.norm() >= 0.3)
        return true;

    return false;
}

void FullSystem::createNewKF()
{
    currentKeyframe = currentFrame;

    currentKeyframe->flagKeyFrame();
    lastTrackedPose = lastTrackedPose * accumulateTransform;
    currentKeyframe->setRawKeyframePose(lastTrackedPose);

    if (mappingEnabled)
    {
        map->addUnprocessedKeyframe(currentKeyframe);
        map->addKeyframePoseRaw(lastTrackedPose);
        map->addFramePose(SE3(), currentKeyframe);
    }

    if (viewerEnabled && viewer)
        viewer->addRawKeyFramePose(lastTrackedPose);

    accumulateTransform = SE3();
}

void FullSystem::resetSystem()
{
    map->clear();
    viewer->resetViewer();
    localMapper->reset();
    lastTrackedPose = SE3(Mat44d::Identity());
    accumulateTransform = SE3(Mat44d::Identity());
    state = SystemState::NotInitialized;
}

size_t FullSystem::getMesh(float *vbuffer, float *nbuffer, size_t bufferSize)
{
    return localMapper->fetchMeshWithNormal(vbuffer, nbuffer);
}

std::vector<SE3> FullSystem::getKeyFramePoseHistory()
{
    return map->getKeyframePoseOptimized();
}

std::vector<SE3> FullSystem::getFramePoseHistory()
{
    return map->getFramePoseOptimized();
}

std::vector<Vec3f> FullSystem::getMapPointPosAll()
{
    return map->getMapPointVec3All();
}

void FullSystem::setMapViewerPtr(MapViewer *viewer)
{
    this->viewer = viewer;
    this->localOptimizer->setViewer(viewer);
}

void FullSystem::setMappingEnable(const bool enable)
{
    mappingEnabled = enable;
}

void FullSystem::setSystemStateToLost()
{
    state = SystemState::Lost;
}

void FullSystem::setGraphMatching(const bool &flag)
{
    useGraphMatching = flag;
}

void FullSystem::setGraphGetNormal(const bool &flag)
{
    shouldCalculateNormal = flag;
}