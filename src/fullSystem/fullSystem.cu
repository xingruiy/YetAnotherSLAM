#include "hip/hip_runtime.h"
#include "fullSystem/fullSystem.h"
#include "denseTracker/cudaImageProc.h"

FullSystem::FullSystem(
    int w, int h,
    Mat33d K,
    int numLvl,
    bool enableViewer)
    : state(SystemState::NotInitialized),
      lastState(SystemState::NotInitialized),
      viewerEnabled(enableViewer),
      mappingEnabled(true),
      imageWidth(w),
      imageHeight(h),
      camIntrinsics(K),
      numProcessedFrames(0)
{
    map = std::make_shared<Map>();
    localOptimizer = std::make_shared<LocalOptimizer>(K, 3, map);
    localMapper = std::make_shared<DenseMapping>(w, h, K);
    loopCloser = std::make_shared<LoopCloser>(K, map);
    coarseTracker = std::make_shared<DenseTracker>(w, h, K, numLvl);

    lastTrackedPose = SE3(Mat44d::Identity());
    accumulateTransform = SE3(Mat44d::Identity());

    bufferVec4wxh.create(h, w, CV_32FC4);
    bufferFloatwxh.create(h, w, CV_32FC1);

    loopThread = std::thread(&LoopCloser::loop, loopCloser.get());
    localOptThread = std::thread(&LocalOptimizer::loop, localOptimizer.get());
}

FullSystem::~FullSystem()
{
    std::cout << "wating other threads to finish..." << std::endl;
    loopCloser->setShouldQuit();
    localOptimizer->setShouldQuit();
    loopThread.join();
    localOptThread.join();
    std::cout << "all threads finished!" << std::endl;
}

void FullSystem::processFrame(Mat rawImage, Mat rawDepth)
{
    rawImage.convertTo(cbufferFloatVec3wxh, CV_32FC3);
    cv::cvtColor(cbufferFloatVec3wxh, cbufferFloatwxh, cv::COLOR_RGB2GRAY);

    currentFrame = std::make_shared<Frame>(
        imageWidth,
        imageHeight,
        camIntrinsics,
        rawImage,
        rawDepth,
        cbufferFloatwxh);

    switch (state)
    {
    case SystemState::NotInitialized:
    {
        if (viewerEnabled && viewer)
            viewer->setCurrentState(-1);

        coarseTracker->setReferenceFrame(currentFrame);
        createNewKF();
        fuseCurrentFrame();
        state = SystemState::OK;

        if (viewerEnabled && viewer)
            viewer->setCurrentState(0);

        break;
    }
    case SystemState::OK:
    {
        auto rval = trackCurrentFrame();
        if (rval)
        {
            fuseCurrentFrame();
            raytraceCurrentFrame();

            if (needNewKF())
            {
                createNewKF();
            }
            else
            {
                map->addFramePose(currentFrame->getTrackingResult(), currentKeyframe);
            }

            if (viewerEnabled && viewer)
                viewer->addTrackingResult(currentFrame->getPoseInLocalMap());
        }
        else
        {
            if (viewerEnabled && viewer)
                viewer->setCurrentState(1);

            state = SystemState::Lost;
        }

        break;
    }
    case SystemState::Lost:

        size_t numAttempted = 0;
        printf("tracking loast, attempt to resuming...\n");
        while (numAttempted <= maxNumRelocAttempt)
        {
            if (tryRelocalizeCurrentFrame(numAttempted > 0))
            {
                if (viewerEnabled && viewer)
                    viewer->setCurrentState(0);

                break;
            }
        }

        break;
    }

    lastState = state;
    if (state == SystemState::OK)
        numProcessedFrames++;
}

bool FullSystem::trackCurrentFrame()
{
    coarseTracker->setTrackingFrame(currentFrame);
    SE3 tRes = coarseTracker->getIncrementalTransform();
    // accumulated local transform
    accumulateTransform = accumulateTransform * tRes.inverse();
    currentFrame->setTrackingResult(accumulateTransform);
    currentFrame->setReferenceKF(currentKeyframe);

    return true;
}

void FullSystem::fuseCurrentFrame()
{

    auto currDepth = coarseTracker->getReferenceDepth();
    localMapper->fuseFrame(currDepth, currentFrame->getPoseInLocalMap());
}

void FullSystem::raytraceCurrentFrame()
{
    localMapper->raytrace(bufferVec4wxh, currentFrame->getPoseInLocalMap());
    coarseTracker->setReferenceInvDepth(bufferVec4wxh);
}

bool FullSystem::tryRelocalizeCurrentFrame(bool updatePoints)
{
    auto matcher = std::make_shared<FeatureMatcher>(PointType::ORB, DescType::ORB);
    currentFrame->detectKeyPoints(matcher);
    Mat descAll;
    const auto desc = map->getPointDescriptorsAll();
    std::vector<std::vector<cv::DMatch>> rawMatches;
    std::vector<cv::DMatch> matches;
    cv::Ptr<cv::DescriptorMatcher> matcher2 = cv::DescriptorMatcher::create(cv::DescriptorMatcher::BRUTEFORCE_HAMMING);
    matcher2->knnMatch(currentFrame->pointDesc, desc, rawMatches, 2);

    for (auto knn : rawMatches)
    {
        if (knn[0].distance / knn[1].distance < 0.8)
            matches.push_back(knn[0]);
    }

    const auto &pts = map->getMapPointsAll();

    std::vector<Vec3f> matchedPoints;
    for (auto m : matches)
    {
        if (pts[m.trainIdx] && !pts[m.trainIdx]->isBad())
            matchedPoints.push_back(pts[m.trainIdx]->getPosWorld().cast<float>());
    }

    if (viewerEnabled && viewer)
        viewer->setMatchedPoints(matchedPoints);

    std::cout << matchedPoints.size() << std::endl;

    return true;
}

bool FullSystem::needNewKF()
{
    auto dt = currentFrame->getTrackingResult();
    Vec3d t = dt.translation();
    if (t.norm() >= 0.1)
        return true;

    Vec3d r = dt.log().tail<3>();
    if (r.norm() >= 0.1)
        return true;

    return false;
}

void FullSystem::createNewKF()
{
    currentKeyframe = currentFrame;

    currentKeyframe->flagKeyFrame();
    lastTrackedPose = lastTrackedPose * accumulateTransform;
    currentKeyframe->setRawKeyframePose(lastTrackedPose);

    if (mappingEnabled)
    {
        map->addUnprocessedKeyframe(currentKeyframe);
        map->setCurrentKeyframe(currentKeyframe);
        map->addKeyframePoseRaw(lastTrackedPose);
        map->addFramePose(SE3(), currentKeyframe);
    }

    if (viewerEnabled && viewer)
        viewer->addRawKeyFramePose(lastTrackedPose);

    accumulateTransform = SE3();
}

void FullSystem::resetSystem()
{

    map->clear();
    viewer->resetViewer();
    localMapper->reset();
    localOptimizer->reset();
    lastTrackedPose = SE3(Mat44d::Identity());
    accumulateTransform = SE3(Mat44d::Identity());
    state = SystemState::NotInitialized;
    lastState = SystemState::NotInitialized;
}

size_t FullSystem::getMesh(float *vbuffer, float *nbuffer, size_t bufferSize)
{
    return localMapper->fetchMeshWithNormal(vbuffer, nbuffer);
}

std::vector<SE3> FullSystem::getKeyFramePoseHistory()
{
    return map->getKeyframePoseOptimized();
}

std::vector<SE3> FullSystem::getFramePoseHistory()
{
    return map->getFramePoseOptimized();
}

std::vector<Vec3f> FullSystem::getMapPointPosAll()
{
    return map->getMapPointVec3All();
}

void FullSystem::setMapViewerPtr(MapViewer *viewer)
{
    this->viewer = viewer;
}

void FullSystem::setMappingEnable(const bool enable)
{
    mappingEnabled = enable;
}

void FullSystem::setSystemStateToLost()
{
    state = SystemState::Lost;
}