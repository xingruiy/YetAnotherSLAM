#include "hip/hip_runtime.h"
#include "fullSystem/fullSystem.h"
#include "denseTracker/cudaImageProc.h"

FullSystem::FullSystem(const char *configFile)
    : viewerEnabled(false)
{
}

FullSystem::FullSystem(
    int w, int h,
    Mat33d K,
    int numLvl,
    bool enableViewer)
    : currentState(-1),
      viewerEnabled(enableViewer)
{
    featureMap = std::make_shared<FeatureMap>(K, 3);
    localMapper = std::make_shared<DenseMapping>(w, h, K);
    coarseTracker = std::make_shared<DenseTracker>(w, h, K, numLvl);

    lastTrackedPose = SE3(Mat44d::Identity());
    accumulateTransform = SE3(Mat44d::Identity());

    bufferVec4wxh.create(h, w, CV_32FC4);
    bufferFloatwxh.create(h, w, CV_32FC1);

    optThread = std::thread(&FeatureMap::optimizationLoop, featureMap.get());
    loopThread = std::thread(&FeatureMap::globalConsistencyLoop, featureMap.get());
}

FullSystem::~FullSystem()
{
    featureMap->setShouldQuit();
    optThread.join();
    loopThread.join();
}

void FullSystem::processFrame(Mat rawImage, Mat rawDepth)
{
    Mat rawImageFloat, rawIntensity;
    rawImage.convertTo(rawImageFloat, CV_32FC3);
    cv::cvtColor(rawImageFloat, rawIntensity, cv::COLOR_RGB2GRAY);
    currentFrame = std::make_shared<Frame>(rawImage, rawDepth, rawIntensity);

    switch (currentState)
    {
    case -1:
    {
        if (viewerEnabled && viewer)
            viewer->setCurrentState(-1);

        coarseTracker->setReferenceFrame(currentFrame);
        createNewKF();
        fuseCurrentFrame();
        currentState = 0;

        if (viewerEnabled && viewer)
            viewer->setCurrentState(0);

        break;
    }
    case 0:
    {
        auto rval = trackCurrentFrame();
        if (rval)
        {
            fuseCurrentFrame();
            raytraceCurrentFrame();

            if (needNewKF())
                createNewKF();
            else
                featureMap->addFrameHistory(currentFrame);

            if (viewerEnabled && viewer)
                viewer->addTrackingResult(currentFrame->getPoseInLocalMap());
        }
        else
        {
            if (viewerEnabled && viewer)
                viewer->setCurrentState(1);

            currentState = 1;
        }

        break;
    }
    case 1:

        size_t numAttempted = 0;
        printf("tracking loast, attempt to resuming...\n");
        while (numAttempted <= maxNumRelocAttempt)
        {
            if (tryRelocalizeCurrentFrame(numAttempted > 0))
            {
                if (viewerEnabled && viewer)
                    viewer->setCurrentState(0);

                break;
            }
        }

        break;
    }
}

bool FullSystem::trackCurrentFrame()
{
    coarseTracker->setTrackingFrame(currentFrame);
    SE3 tRes = coarseTracker->getIncrementalTransform();
    // accumulated local transform
    accumulateTransform = accumulateTransform * tRes.inverse();
    currentFrame->setTrackingResult(accumulateTransform);
    currentFrame->setReferenceKF(referenceFrame);

    return true;
}

void FullSystem::fuseCurrentFrame()
{
    auto currDepth = coarseTracker->getReferenceDepth();
    localMapper->fuseFrame(currDepth, currentFrame->getPoseInLocalMap());
}

void FullSystem::raytraceCurrentFrame()
{
    localMapper->raytrace(bufferVec4wxh, currentFrame->getPoseInLocalMap());
    coarseTracker->setReferenceInvDepth(bufferVec4wxh);
}

bool FullSystem::tryRelocalizeCurrentFrame(bool updatePoints)
{
    return true;
}

bool FullSystem::needNewKF()
{
    auto dt = currentFrame->getTrackingResult();
    Vec3d t = dt.translation();
    if (t.norm() >= 0.1)
        return true;

    Vec3d r = dt.log().tail<3>();
    if (r.norm() >= 0.1)
        return true;

    return false;
}

void FullSystem::createNewKF()
{
    referenceFrame = currentFrame;

    /* 
        Flag for keyframe
    */
    referenceFrame->flagKeyFrame();
    lastTrackedPose = lastTrackedPose * accumulateTransform;
    referenceFrame->setRawKeyframePose(lastTrackedPose);
    featureMap->addReferenceFrame(referenceFrame);

    rawKeyFramePoseHistory.push_back(lastTrackedPose);
    accumulateTransform = SE3();
}

void FullSystem::resetSystem()
{
    currentState = -1;
    localMapper->reset();
    featureMap->reset();
    rawFramePoseHistory.clear();
    rawKeyFramePoseHistory.clear();

    lastTrackedPose = SE3(Mat44d::Identity());
    accumulateTransform = SE3(Mat44d::Identity());
}

std::vector<SE3> FullSystem::getRawFramePoseHistory() const
{
    return rawFramePoseHistory;
}

std::vector<SE3> FullSystem::getRawKeyFramePoseHistory() const
{
    return rawKeyFramePoseHistory;
}

size_t FullSystem::getMesh(float *vbuffer, float *nbuffer, size_t bufferSize)
{
    return localMapper->fetch_mesh_with_normal(vbuffer, nbuffer);
}

std::vector<SE3> FullSystem::getKeyFramePoseHistory()
{
    return featureMap->getKeyFrameHistory();
}

std::vector<SE3> FullSystem::getFramePoseHistory()
{
    return featureMap->getFrameHistory();
}

std::vector<Vec3f> FullSystem::getActiveKeyPoints()
{
    return featureMap->getActivePoints();
}

std::vector<Vec3f> FullSystem::getStableKeyPoints()
{
    return featureMap->getStablePoints();
}

void FullSystem::setMapViewerPtr(MapViewer *viewer)
{
    this->viewer = viewer;
}