#include "hip/hip_runtime.h"
#include "RGBDTracking.h"
#include "TrackingUtils.h"
#include "ImageProc.h"

RGBDTracking::RGBDTracking(const int &w, const int &h,
                           const Eigen::Matrix3d &K,
                           const bool &bRGB, const bool &bDepth)
{
    if (bRGB && bDepth)
        mModal = TrackingModal::RGB_AND_DEPTH;
    else if (bRGB)
        mModal = TrackingModal::RGB_ONLY;
    else
        mModal = TrackingModal::DEPTH_ONLY;

    for (int lvl = 0; lvl < NUM_PYR; ++lvl)
    {
        int wLvl = w >> lvl;
        int hLvl = h >> lvl;

        mK[lvl] = K / (1 << lvl);
        mK[lvl](2, 2) = 1.0f;

        mvWidth[lvl] = wLvl;
        mvHeight[lvl] = hLvl;
    }

    for (int lvl = 0; lvl < NUM_PYR; ++lvl)
    {
        int wLvl = mvWidth[lvl];
        int hLvl = mvHeight[lvl];

        mvCurrentDepth[lvl].create(hLvl, wLvl, CV_32FC1);
        mvReferenceDepth[lvl].create(hLvl, wLvl, CV_32FC1);
        mvCurrentIntensity[lvl].create(hLvl, wLvl, CV_32FC1);
        mvReferenceIntensity[lvl].create(hLvl, wLvl, CV_32FC1);
        mvIntensityGradientX[lvl].create(hLvl, wLvl, CV_32FC1);
        mvIntensityGradientY[lvl].create(hLvl, wLvl, CV_32FC1);
        mvReferencePointTransformed[lvl].create(hLvl, wLvl, CV_32FC4);

        mvCurrentInvDepth[lvl].create(hLvl, wLvl, CV_32FC1);
        mvReferenceInvDepth[lvl].create(hLvl, wLvl, CV_32FC1);
        mvInvDepthGradientX[lvl].create(hLvl, wLvl, CV_32FC1);
        mvInvDepthGradientY[lvl].create(hLvl, wLvl, CV_32FC1);

        mvCurrentVMap[lvl].create(hLvl, wLvl, CV_32FC4);
        mvCurrentNMap[lvl].create(hLvl, wLvl, CV_32FC4);
        mvReferenceVMap[lvl].create(hLvl, wLvl, CV_32FC4);
        mvReferenceNMap[lvl].create(hLvl, wLvl, CV_32FC4);
    }

    // Create temporary buffers
    mGpuBufferFloat96x29.create(96, 29, CV_32FC1);
    mGpuBufferFloat96x3.create(96, 3, CV_32FC1);
    mGpuBufferFloat96x2.create(96, 2, CV_32FC1);
    mGpuBufferFloat96x1.create(96, 1, CV_32FC1);
    mGpuBufferFloat1x29.create(1, 29, CV_32FC1);
    mGpuBufferFloat1x3.create(1, 2, CV_32FC1);
    mGpuBufferFloat1x2.create(1, 2, CV_32FC1);
    mGpuBufferFloat1x1.create(1, 1, CV_32FC1);
    mGpuBufferVector4HxW.create(h, w, CV_32FC4);
    mGpuBufferVector7HxW.create(h, w, CV_32FC(7));
    mGpuBufferRawDepth.create(h, w, CV_32FC1);
}

void RGBDTracking::SetReferenceImage(const cv::Mat &imGray)
{
    cv::Mat imGrayFloat;
    imGray.convertTo(imGrayFloat, CV_32FC1);

    for (int lvl = 0; lvl < NUM_PYR; ++lvl)
    {
        if (lvl == 0)
            mvReferenceIntensity[0].upload(imGrayFloat);
        else
            cv::cuda::pyrDown(mvReferenceIntensity[lvl - 1], mvReferenceIntensity[lvl]);
    }
}

void RGBDTracking::SetReferenceDepth(const cv::Mat &imDepth)
{
    for (int lvl = 0; lvl < NUM_PYR; ++lvl)
    {
        if (lvl == 0)
        {
            mGpuBufferRawDepth.upload(imDepth);
            DepthToInvDepth(mGpuBufferRawDepth, mvReferenceInvDepth[lvl]);
        }
        else
            PyrDownDepth(mvReferenceInvDepth[lvl - 1], mvReferenceInvDepth[lvl]);

        float invfx = 1.0 / mK[lvl](0, 0);
        float invfy = 1.0 / mK[lvl](1, 1);
        float cx = mK[lvl](0, 2);
        float cy = mK[lvl](1, 2);

        ComputeVertexMap(mvReferenceInvDepth[lvl], mvReferenceVMap[lvl], invfx, invfy, cx, cy, 3.0f);
        ComputeNormalMap(mvReferenceVMap[lvl], mvReferenceNMap[lvl]);
    }
}

void RGBDTracking::SetTrackingImage(const cv::Mat &imGray)
{
    cv::Mat imGrayFloat;
    imGray.convertTo(imGrayFloat, CV_32FC1);

    for (int lvl = 0; lvl < NUM_PYR; ++lvl)
    {
        if (lvl == 0)
            mvCurrentIntensity[lvl].upload(imGrayFloat);
        else
            cv::cuda::pyrDown(mvCurrentIntensity[lvl - 1], mvCurrentIntensity[lvl]);

        ComputeImageGradientCentralDifference(mvCurrentIntensity[lvl], mvIntensityGradientX[lvl], mvIntensityGradientY[lvl]);
    }
}

void RGBDTracking::SetTrackingDepth(const cv::Mat &imDepth)
{
    for (int lvl = 0; lvl < NUM_PYR; ++lvl)
    {
        if (lvl == 0)
        {
            mGpuBufferRawDepth.upload(imDepth);
            DepthToInvDepth(mGpuBufferRawDepth, mvCurrentInvDepth[lvl]);
        }
        else
            PyrDownDepth(mvCurrentInvDepth[lvl - 1], mvCurrentInvDepth[lvl]);

        // ComputeImageGradientCentralDifference(mvCurrentInvDepth[lvl], mvInvDepthGradientX[lvl], mvInvDepthGradientY[lvl]);

        float invfx = 1.0 / mK[lvl](0, 0);
        float invfy = 1.0 / mK[lvl](1, 1);
        float cx = mK[lvl](0, 2);
        float cy = mK[lvl](1, 2);

        ComputeVertexMap(mvCurrentInvDepth[lvl], mvCurrentVMap[lvl], invfx, invfy, cx, cy, 3.0f);
        ComputeNormalMap(mvCurrentVMap[lvl], mvCurrentNMap[lvl]);
    }

    // cv::Mat vmap(mvCurrentVMap[0]);
    // cv::Mat nmap(mvCurrentNMap[0]);
    // cv::imshow("vmap", vmap);
    // cv::imshow("nmap", nmap);
    // cv::waitKey(0);
}

void RGBDTracking::SetReferenceMap(const cv::cuda::GpuMat vmap)
{
    vmap.copyTo(mvReferenceVMap[0]);
    for (int lvl = 0; lvl < NUM_PYR; ++lvl)
    {
        if (lvl != 0)
            PyrDownDepth(mvReferenceVMap[lvl - 1], mvReferenceVMap[lvl]);

        ComputeNormalMap(mvReferenceVMap[lvl], mvReferenceNMap[lvl]);
    }
}

Sophus::SE3d RGBDTracking::GetTransform(const Sophus::SE3d &init, const bool bSwapBuffer)
{
    int nIteration = 0;
    int nSuccessfulIteration = 0;

    Sophus::SE3d estimate = init;
    Sophus::SE3d lastSuccessEstimate = estimate;
    std::vector<int> vIterations = {10, 5, 3, 3, 3};

    for (int lvl = NUM_PYR - 1; lvl >= 0; --lvl)
    {
        float lastError = std::numeric_limits<float>::max();
        for (int iter = 0; iter < vIterations[lvl]; ++iter)
        {
            Eigen::Matrix<float, 6, 6> hessian = Eigen::Matrix<float, 6, 6>::Zero();
            Eigen::Matrix<float, 6, 1> residual = Eigen::Matrix<float, 6, 1>::Zero();

            switch (mModal)
            {
            case TrackingModal::RGB_ONLY:
                ComputeSingleStepRGB(lvl, estimate, hessian.data(), residual.data());
                break;

            case TrackingModal::DEPTH_ONLY:
                ComputeSingleStepDepth(lvl, estimate, hessian.data(), residual.data());
                break;

            case TrackingModal::RGB_AND_DEPTH:
                // ComputeSingleStepRGBD(lvl, estimate, hessian.data(), residual.data());
                ComputeSingleStepRGBDLinear(lvl, estimate, hessian.data(), residual.data());
                break;
            }

            float error = sqrt(residualSum) / (numResidual + 6);
            Eigen::Matrix<double, 6, 1> update = hessian.cast<double>().ldlt().solve(residual.cast<double>());

            if (std::isnan(update(0)))
            {
                mbTrackingGood = false;
                return Sophus::SE3d();
            }

            // update = ClampEigenVector(update, 0.05, -0.05);

            estimate = Sophus::SE3d::exp(update) * estimate;
            if (error < lastError)
            {
                lastSuccessEstimate = estimate;
                lastError = error;
                nSuccessfulIteration++;
            }

            nIteration++;
        }
    }

    if (bSwapBuffer)
    {
        SwapFrameBuffer();
    }

    mbTrackingGood = true;
    return lastSuccessEstimate;
}

void RGBDTracking::TransformReferencePoint(const int lvl, const Sophus::SE3d &T)
{
    auto refInvDepth = mvReferenceInvDepth[lvl];
    auto refPtTransformedLvl = mvReferencePointTransformed[lvl];
    auto KLvl = mK[lvl];

    ::TransformReferencePoint(refInvDepth, refPtTransformedLvl, KLvl, T);
}

void RGBDTracking::ComputeSingleStepRGB(
    const int lvl,
    const Sophus::SE3d &T,
    float *hessian,
    float *residual)
{
    TransformReferencePoint(lvl, T);

    const int w = mvWidth[lvl];
    const int h = mvHeight[lvl];

    se3StepRGBResidualFunctor functor;
    functor.w = w;
    functor.h = h;
    functor.n = w * h;
    functor.refInt = mvReferenceIntensity[lvl];
    functor.currInt = mvCurrentIntensity[lvl];
    functor.currGx = mvIntensityGradientX[lvl];
    functor.currGy = mvIntensityGradientY[lvl];
    functor.refPtWarped = mvReferencePointTransformed[lvl];
    functor.refResidual = mGpuBufferVector4HxW;
    functor.fx = mK[lvl](0, 0);
    functor.fy = mK[lvl](1, 1);
    functor.cx = mK[lvl](0, 2);
    functor.cy = mK[lvl](1, 2);
    functor.out = mGpuBufferFloat96x2;

    callDeviceFunctor<<<96, 224>>>(functor);
    cv::cuda::reduce(mGpuBufferFloat96x2, mGpuBufferFloat1x2, 0, cv::REDUCE_SUM);
    cv::Mat hostData(mGpuBufferFloat1x2);

    iResidualSum = hostData.ptr<float>(0)[0];
    numResidual = hostData.ptr<float>(0)[1];

    VarianceEstimator estimator;
    estimator.w = w;
    estimator.h = h;
    estimator.n = w * h;
    estimator.meanEstimated = iResidualSum / numResidual;
    estimator.residual = mGpuBufferVector4HxW;
    estimator.out = mGpuBufferFloat96x1;

    callDeviceFunctor<<<96, 224>>>(estimator);
    cv::cuda::reduce(mGpuBufferFloat96x1, mGpuBufferFloat1x1, 0, cv::REDUCE_SUM);
    mGpuBufferFloat1x1.download(hostData);

    float squaredDeviationSum = hostData.ptr<float>(0)[0];
    float varEstimated = sqrt(squaredDeviationSum / (numResidual - 1));

    se3StepRGBFunctor sfunctor;
    sfunctor.w = w;
    sfunctor.h = h;
    sfunctor.n = w * h;
    sfunctor.huberTh = 4.685 * varEstimated;
    sfunctor.refPtWarped = mvReferencePointTransformed[lvl];
    sfunctor.refResidual = mGpuBufferVector4HxW;
    sfunctor.fx = mK[lvl](0, 0);
    sfunctor.fy = mK[lvl](1, 1);
    sfunctor.out = mGpuBufferFloat96x29;

    callDeviceFunctor<<<96, 224>>>(sfunctor);
    cv::cuda::reduce(mGpuBufferFloat96x29, mGpuBufferFloat1x29, 0, cv::REDUCE_SUM);

    mGpuBufferFloat1x29.download(hostData);
    RankUpdateHessian<6, 7>(hostData.ptr<float>(0), hessian, residual);

    residualSum = hostData.ptr<float>(0)[27];
}

void RGBDTracking::SwapFrameBuffer()
{
    for (int lvl = 0; lvl < NUM_PYR; ++lvl)
    {
        std::swap(mvReferenceVMap[lvl], mvCurrentVMap[lvl]);
        std::swap(mvReferenceNMap[lvl], mvCurrentNMap[lvl]);
        std::swap(mvReferenceInvDepth[lvl], mvCurrentInvDepth[lvl]);
        std::swap(mvReferenceIntensity[lvl], mvCurrentIntensity[lvl]);
    }
}

struct Point2PlaneICPFunctor
{
    cv::cuda::PtrStep<Eigen::Vector4f> vmap_curr;
    cv::cuda::PtrStep<Eigen::Vector4f> nmap_curr;
    cv::cuda::PtrStep<Eigen::Vector4f> vmap_last;
    cv::cuda::PtrStep<Eigen::Vector4f> nmap_last;
    int cols, rows, N;
    float fx, fy, cx, cy;
    float angleTH, distTH;
    Sophus::SE3f T_last_curr;
    mutable cv::cuda::PtrStep<float> out;

    __device__ __forceinline__ bool ProjectPoint(int &x, int &y,
                                                 Eigen::Vector3f &v_curr,
                                                 Eigen::Vector3f &n_last,
                                                 Eigen::Vector3f &v_last) const;
    __device__ __forceinline__ void GetProduct(int &k, float *out) const;
    __device__ __forceinline__ void operator()() const;
};

__device__ __forceinline__ bool Point2PlaneICPFunctor::ProjectPoint(int &x, int &y,
                                                                    Eigen::Vector3f &v_curr,
                                                                    Eigen::Vector3f &n_last,
                                                                    Eigen::Vector3f &v_last) const
{
    Eigen::Vector4f v_last_c = vmap_last.ptr(y)[x];
    if (v_last_c(3) < 0)
        return false;

    v_last = T_last_curr * v_last_c.head<3>();

    float invz = 1.0 / v_last(2);
    int u = __float2int_rd(fx * v_last(0) * invz + cx + 0.5);
    int v = __float2int_rd(fy * v_last(1) * invz + cy + 0.5);
    if (u < 0 || v < 0 || u >= cols || v >= rows)
        return false;

    Eigen::Vector4f v_curr_c = vmap_curr.ptr(v)[u];
    v_curr = v_curr_c.head<3>();
    if (v_curr_c(3) < 0)
        return false;

    Eigen::Vector4f n_last_c = nmap_last.ptr(y)[x];
    n_last = T_last_curr.so3() * n_last_c.head<3>();

    Eigen::Vector4f n_curr_c = nmap_curr.ptr(v)[u];

    float dist = (v_last - v_curr).norm();
    float angle = n_curr_c.head<3>().cross(n_last).norm();

    return (angle < angleTH && dist <= distTH && n_last_c(3) > 0 && n_curr_c(3) > 0);
}

__device__ __forceinline__ void Point2PlaneICPFunctor::GetProduct(int &k, float *sum) const
{
    int y = k / cols;
    int x = k - (y * cols);

    Eigen::Vector3f v_curr, n_last, v_last;
    float row[7] = {0, 0, 0, 0, 0, 0, 0};
    bool found = ProjectPoint(x, y, v_curr, n_last, v_last);

    if (found)
    {
        *(Eigen::Vector3f *)&row[0] = n_last;
        *(Eigen::Vector3f *)&row[3] = v_last.cross(n_last);
        row[6] = n_last.dot(v_curr - v_last);
    }

    int count = 0;
#pragma unroll
    for (int i = 0; i < 7; ++i)
#pragma unroll
        for (int j = i; j < 7; ++j)
            sum[count++] = row[i] * row[j];
    sum[count] = (float)found;
}

__device__ __forceinline__ void Point2PlaneICPFunctor::operator()() const
{
    float sum[29] = {0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0};
    float val[29];
    for (int k = blockIdx.x * blockDim.x + threadIdx.x; k < N; k += blockDim.x * gridDim.x)
    {
        GetProduct(k, val);

#pragma unroll
        for (int i = 0; i < 29; ++i)
        {
            sum[i] += val[i];
        }
    }

    BlockReduceSum<float, 29>(sum);

    if (threadIdx.x == 0)
    {
#pragma unroll
        for (int i = 0; i < 29; ++i)
            out.ptr(blockIdx.x)[i] = sum[i];
    }
}

void RGBDTracking::ComputeSingleStepDepth(
    const int lvl,
    const Sophus::SE3d &T,
    float *hessian,
    float *residual)
{
    int cols = mvWidth[lvl];
    int rows = mvHeight[lvl];

    Point2PlaneICPFunctor P2PIcpFunctor;
    P2PIcpFunctor.out = mGpuBufferFloat96x29;
    P2PIcpFunctor.vmap_curr = mvCurrentVMap[lvl];
    P2PIcpFunctor.nmap_curr = mvCurrentNMap[lvl];
    P2PIcpFunctor.vmap_last = mvReferenceVMap[lvl];
    P2PIcpFunctor.nmap_last = mvReferenceNMap[lvl];
    P2PIcpFunctor.cols = cols;
    P2PIcpFunctor.rows = rows;
    P2PIcpFunctor.N = cols * rows;
    P2PIcpFunctor.T_last_curr = T.cast<float>();
    P2PIcpFunctor.angleTH = sin(20.f * 3.14159254f / 180.f);
    P2PIcpFunctor.distTH = 0.01;
    P2PIcpFunctor.fx = mK[lvl](0, 0);
    P2PIcpFunctor.fy = mK[lvl](1, 1);
    P2PIcpFunctor.cx = mK[lvl](0, 2);
    P2PIcpFunctor.cy = mK[lvl](1, 2);

    callDeviceFunctor<<<96, 224>>>(P2PIcpFunctor);
    cv::cuda::reduce(mGpuBufferFloat96x29, mGpuBufferFloat1x29, 0, cv::REDUCE_SUM);

    cv::Mat hostData(mGpuBufferFloat1x29);
    RankUpdateHessian<6, 7>(hostData.ptr<float>(0), hessian, residual);

    residualSum = hostData.ptr<float>(0)[27];
}

void RGBDTracking::ComputeSingleStepRGBD(
    const int lvl,
    const Sophus::SE3d &T,
    float *hessian,
    float *residual)
{
    TransformReferencePoint(lvl, T);

    const int w = mvWidth[lvl];
    const int h = mvHeight[lvl];

    se3StepRGBDResidualFunctor functor;
    functor.w = w;
    functor.h = h;
    functor.n = w * h;
    functor.refInt = mvReferenceIntensity[lvl];
    functor.currInt = mvCurrentIntensity[lvl];
    functor.currGx = mvIntensityGradientX[lvl];
    functor.currGy = mvIntensityGradientY[lvl];
    functor.currInvDepth = mvCurrentInvDepth[lvl];
    functor.currInvDepthGx = mvInvDepthGradientX[lvl];
    functor.currInvDepthGy = mvInvDepthGradientY[lvl];
    functor.refPtWarped = mvReferencePointTransformed[lvl];
    functor.refResidual = mGpuBufferVector7HxW;
    functor.fx = mK[lvl](0, 0);
    functor.fy = mK[lvl](1, 1);
    functor.cx = mK[lvl](0, 2);
    functor.cy = mK[lvl](1, 2);
    functor.out = mGpuBufferFloat96x3;

    callDeviceFunctor<<<96, 224>>>(functor);
    cv::cuda::reduce(mGpuBufferFloat96x3, mGpuBufferFloat1x3, 0, cv::REDUCE_SUM);
    cv::Mat hostData(mGpuBufferFloat1x3);

    float iResidualSum = hostData.ptr<float>(0)[0];
    float dResidualSum = hostData.ptr<float>(0)[1];
    numResidual = hostData.ptr<float>(0)[2];

    VarCov2DEstimator estimator;
    estimator.h = h;
    estimator.w = w;
    estimator.n = h * w;
    estimator.meanEstimated = Eigen::Vector2f(iResidualSum, dResidualSum) / numResidual;
    estimator.residual = mGpuBufferVector7HxW;
    estimator.out = mGpuBufferFloat96x3;

    callDeviceFunctor<<<96, 224>>>(estimator);
    cv::cuda::reduce(mGpuBufferFloat96x3, mGpuBufferFloat1x3, 0, cv::REDUCE_SUM);
    mGpuBufferFloat1x3.download(hostData);

    Eigen::Matrix2f varEstimated;
    varEstimated(0, 0) = hostData.ptr<float>(0)[0];
    varEstimated(1, 1) = hostData.ptr<float>(0)[1];
    varEstimated(0, 1) = varEstimated(1, 0) = hostData.ptr<float>(0)[2];
    varEstimated /= (numResidual - 1);

    se3StepRGBDFunctor sfunctor;
    sfunctor.w = w;
    sfunctor.h = h;
    sfunctor.n = w * h;
    sfunctor.stddevI = 1.345 * varEstimated(0, 0);
    sfunctor.stddevD = 4.685 * varEstimated(1, 1);
    sfunctor.precision = varEstimated.inverse();
    sfunctor.refPtWarped = mvReferencePointTransformed[lvl];
    sfunctor.refResidual = mGpuBufferVector7HxW;
    sfunctor.fx = mK[lvl](0, 0);
    sfunctor.fy = mK[lvl](1, 1);
    sfunctor.out = mGpuBufferFloat96x29;

    callDeviceFunctor<<<96, 224>>>(sfunctor);
    cv::cuda::reduce(mGpuBufferFloat96x29, mGpuBufferFloat1x29, 0, cv::REDUCE_SUM);

    mGpuBufferFloat1x29.download(hostData);
    RankUpdateHessian<6, 7>(hostData.ptr<float>(0), hessian, residual);

    residualSum = hostData.ptr<float>(0)[27];
}

void RGBDTracking::ComputeSingleStepRGBDLinear(
    const int lvl,
    const Sophus::SE3d &T,
    float *hessian,
    float *residual)
{
    Eigen::Map<Eigen::Matrix<float, 6, 6>> hessianMapped(hessian);
    Eigen::Map<Eigen::Matrix<float, 6, 1>> residualMapped(residual);

    Eigen::Matrix<float, 6, 6> hessianBuffer;
    Eigen::Matrix<float, 6, 1> residualBuffer;

    ComputeSingleStepRGB(lvl, T, hessianBuffer.data(), residualBuffer.data());

    hessianMapped += hessianBuffer;
    residualMapped += residualBuffer;

    hessianBuffer.setZero();
    residualBuffer.setZero();

    ComputeSingleStepDepth(lvl, T, hessianBuffer.data(), residualBuffer.data());

    hessianMapped += 100 * hessianBuffer;
    residualMapped += 10 * residualBuffer;
}

cv::cuda::GpuMat RGBDTracking::GetReferenceDepth(const int lvl) const
{
    return mGpuBufferRawDepth;
}

bool RGBDTracking::IsTrackingGood() const
{
    return mbTrackingGood;
}
