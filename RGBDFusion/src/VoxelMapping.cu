#include "MappingUtils.h"
#include "VoxelMapping.h"

VoxelMapping::VoxelMapping(const int w, const int h, const Eigen::Matrix3f &K)
    : mK(K.cast<double>())
{
  deviceMap.create(80000, 40000, 40000, 0.006f, 0.02f);
  deviceMap.reset();
  zRangeX.create(h / 8, w / 8, CV_32FC1);
  zRangeY.create(h / 8, w / 8, CV_32FC1);

  hipMalloc((void **)&mplRenderingBlock, sizeof(RenderingBlock) * 100000);
}

VoxelMapping::~VoxelMapping()
{
  deviceMap.release();
  SafeCall(hipFree((void **)&mplRenderingBlock));
}

void VoxelMapping::FuseFrame(cv::cuda::GpuMat depth, const Sophus::SE3d &T)
{
  mNumVisibleBlocks = 0;

  ::fuseDepth(
      deviceMap,
      depth,
      T,
      mK,
      mNumVisibleBlocks);
}

void VoxelMapping::TracingDepth(cv::cuda::GpuMat &vertex, const Sophus::SE3d &T)
{
  if (mNumVisibleBlocks == 0)
    return;

  ::create_rendering_blocks(
      deviceMap,
      mNumVisibleBlocks,
      mNumRenderingBlocks,
      zRangeX,
      zRangeY,
      mplRenderingBlock,
      T,
      mK);

  if (mNumRenderingBlocks != 0)
  {

    ::raycast(
        deviceMap,
        vertex,
        zRangeX,
        zRangeY,
        T,
        mK);
  }
}

void VoxelMapping::reset()
{
  deviceMap.reset();
}

size_t VoxelMapping::fetchMeshWithNormal(void *vertex, void *normal)
{
  uint count_triangle = 0;

  ::create_mesh_with_normal(
      deviceMap,
      mNumVisibleBlocks,
      count_triangle,
      vertex,
      normal);

  return (size_t)count_triangle;
}
