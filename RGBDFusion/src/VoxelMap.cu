#include "hip/hip_runtime.h"
#include "VoxelMap.h"
#include <fstream>

__global__ void resetHashKernel(HashEntry *mplHashTable, int numEntry)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numEntry)
        return;

    mplHashTable[index].ptr = -1;
    mplHashTable[index].offset = -1;
}

__global__ void resetHeapKernel(int *mplHeap, int *mplHeapPtr, int numBlock)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numBlock)
        return;

    if (index == 0)
        mplHeapPtr[0] = numBlock - 1;

    mplHeap[index] = numBlock - index - 1;
}

void MapStruct::reset()
{
    dim3 block(1024);
    dim3 grid(cv::divUp(hashTableSize, block.x));
    resetHashKernel<<<grid, block>>>(mplHashTable, hashTableSize);

    grid = dim3(cv::divUp(voxelBlockSize, block.x));
    resetHeapKernel<<<grid, block>>>(mplHeap, mplHeapPtr, voxelBlockSize);

    hipMemset(mpLinkedListHead, 0, sizeof(int));
    hipMemset(mplBucketMutex, 0, sizeof(int) * bucketSize);
    hipMemset(mplVoxelBlocks, 0, sizeof(Voxel) * BlockSize3 * voxelBlockSize);
}

void MapStruct::create(
    int hashTableSize,
    int bucketSize,
    int voxelBlockSize,
    float voxelSize,
    float truncationDist)
{
    hipMalloc((void **)&mpLinkedListHead, sizeof(int));
    hipMalloc((void **)&mplHeapPtr, sizeof(int));
    hipMalloc((void **)&visibleBlockNum, sizeof(uint));
    hipMalloc((void **)&mplBucketMutex, sizeof(int) * bucketSize);
    hipMalloc((void **)&mplHeap, sizeof(int) * voxelBlockSize);
    hipMalloc((void **)&mplHashTable, sizeof(HashEntry) * hashTableSize);
    hipMalloc((void **)&visibleTable, sizeof(HashEntry) * hashTableSize);
    hipMalloc((void **)&mplVoxelBlocks, sizeof(Voxel) * voxelBlockSize * BlockSize3);

    this->hashTableSize = hashTableSize;
    this->bucketSize = bucketSize;
    this->voxelBlockSize = voxelBlockSize;
    this->voxelSize = voxelSize;
    this->truncationDist = truncationDist;
}

MapStruct::MapStruct(int SizeInMB)
{
    int nHashEntry = 0;  // 160kb
    int nVoxelBlock = 0; // 3072kb
    int nBucket = 0;
    float voxelSize = 0.005;
    float TruncationDist = 0.02;
}

void MapStruct::release()
{
    hipFree((void *)mplHeap);
    hipFree((void *)mplHeapPtr);
    hipFree((void *)mplHashTable);
    hipFree((void *)mplBucketMutex);
    hipFree((void *)mpLinkedListHead);
    hipFree((void *)mplVoxelBlocks);
    hipFree((void *)visibleBlockNum);
    hipFree((void *)visibleTable);
}

void MapStruct::getVisibleBlockCount(uint &hostData)
{
    hipMemcpy(&hostData, visibleBlockNum, sizeof(uint), hipMemcpyDeviceToHost);
}

void MapStruct::resetVisibleBlockCount()
{
    hipMemset(visibleBlockNum, 0, sizeof(uint));
}

bool MapStruct::empty()
{
    return bucketSize == 0;
}

void MapStruct::UpdateMesh()
{
    if (!mbHasMesh && mpMeshEngine)
    {
        mpMeshEngine->Meshify(this);
        SafeCall(hipDeviceSynchronize());
        SafeCall(hipGetLastError());

        mbHasMesh = true;
    }
}

void MapStruct::setMeshEngine(MeshEngine *pMeshEngine)
{
    mpMeshEngine = pMeshEngine;
}
