#include "hip/hip_runtime.h"
#include "VoxelMap.h"
#include "CudaUtils.h"
#include "ParallelScan.h"
#include "VoxelStructUtils.h"

#define HASHENTRY_IN_BYTE 0.00002
#define VOXEL_BLOCK_IN_BYTE 0.000003

long unsigned int MapStruct::nNextId = 0;

__global__ void ResetHash_kernel(HashEntry *mplHashTable, int numEntry)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numEntry)
        return;

    mplHashTable[index].ptr = -1;
    mplHashTable[index].offset = -1;
}

__global__ void ResetHeap_kernel(int *mplHeap, int *mplHeapPtr, int numBlock)
{
    int index = threadIdx.x + blockDim.x * blockIdx.x;
    if (index >= numBlock)
        return;

    if (index == 0)
        mplHeapPtr[0] = numBlock - 1;

    mplHeap[index] = numBlock - index - 1;
}

void MapStruct::Reset()
{
    dim3 block(1024);
    dim3 grid(cv::divUp(hashTableSize, block.x));
    ResetHash_kernel<<<grid, block>>>(mplHashTable, hashTableSize);

    grid = dim3(cv::divUp(voxelBlockSize, block.x));
    ResetHeap_kernel<<<grid, block>>>(mplHeap, mplHeapPtr, voxelBlockSize);

    hipMemset(mpLinkedListHead, 0, sizeof(int));
    hipMemset(mplBucketMutex, 0, sizeof(int) * bucketSize);
    hipMemset(mplVoxelBlocks, 0, sizeof(Voxel) * BlockSize3 * voxelBlockSize);
}

void MapStruct::create(
    int hashTableSize,
    int bucketSize,
    int voxelBlockSize,
    float voxelSize,
    float truncationDist)
{
    SafeCall(hipMalloc((void **)&mpLinkedListHead, sizeof(int)));
    SafeCall(hipMalloc((void **)&mplHeapPtr, sizeof(int)));
    SafeCall(hipMalloc((void **)&visibleBlockNum, sizeof(uint)));
    SafeCall(hipMalloc((void **)&mplBucketMutex, sizeof(int) * bucketSize));
    SafeCall(hipMalloc((void **)&mplHeap, sizeof(int) * voxelBlockSize));
    SafeCall(hipMalloc((void **)&mplHashTable, sizeof(HashEntry) * hashTableSize));
    SafeCall(hipMalloc((void **)&visibleTable, sizeof(HashEntry) * hashTableSize));
    SafeCall(hipMalloc((void **)&mplVoxelBlocks, sizeof(Voxel) * voxelBlockSize * BlockSize3));

    this->hashTableSize = hashTableSize;
    this->bucketSize = bucketSize;
    this->voxelBlockSize = voxelBlockSize;
    this->voxelSize = voxelSize;
    this->truncationDist = truncationDist;
}

MapStruct::MapStruct(const Eigen::Matrix3f &K)
    : mFootPrintInMB(0), mbInHibernation(false), mbActive(true),
      mbHasMesh(false), mpMeshEngine(NULL), mplHeap(NULL),
      mplHeapPtr(NULL), mplBucketMutex(NULL), mplHashTable(NULL),
      mplVoxelBlocks(NULL), mpLinkedListHead(NULL), mK(K),
      mbVertexBufferCreated(false)
{
    // Get a random colour taint for visualization
    mColourTaint = 255 * rand() / (double)RAND_MAX;
    mnId = nNextId++;
}

MapStruct::MapStruct(int SizeInMB)
{
    // int nHashEntry = 0;  // 160kb
    // int nVoxelBlock = 0; // 3072kb
    // int nBucket = 0;
    // float voxelSize = 0.005;
    // float TruncationDist = 0.02;
}

void MapStruct::Release()
{
    SafeCall(hipFree((void *)mplHeap));
    SafeCall(hipFree((void *)mplHeapPtr));
    SafeCall(hipFree((void *)mplHashTable));
    SafeCall(hipFree((void *)mplBucketMutex));
    SafeCall(hipFree((void *)mpLinkedListHead));
    SafeCall(hipFree((void *)mplVoxelBlocks));
    SafeCall(hipFree((void *)visibleBlockNum));
    SafeCall(hipFree((void *)visibleTable));

    if (mbHasMesh && N > 0)
    {
        N = 0;
        free(mplPoint);
        free(mplNormal);
    }

    mplHeap = NULL;
    mplHeapPtr = NULL;
    mplHashTable = NULL;
    visibleTable = NULL;
    mplVoxelBlocks = NULL;
    mplBucketMutex = NULL;
    visibleBlockNum = NULL;
    mpLinkedListHead = NULL;
    mbInHibernation = false;
    mFootPrintInMB = 0;
}

bool MapStruct::empty()
{
    return bucketSize == 0;
}

void MapStruct::GenerateMesh()
{
    if (!mbHasMesh && mpMeshEngine && !mbInHibernation)
    {
        mpMeshEngine->Meshify(this);
        SafeCall(hipDeviceSynchronize());
        SafeCall(hipGetLastError());

        mbHasMesh = true;
    }
}

void MapStruct::DeleteMesh()
{
    if (mbHasMesh)
    {
        N = 0;
        delete mplPoint;
        delete mplNormal;
        mbHasMesh = false;
    }
}

void MapStruct::SetMeshEngine(MeshEngine *pMeshEngine)
{
    mpMeshEngine = pMeshEngine;
}

void MapStruct::SetRayTraceEngine(RayTraceEngine *pRayTraceEngine)
{
    mpRayTraceEngine = pRayTraceEngine;
}

void MapStruct::Swap(MapStruct *pMapStruct)
{
    {
        using std::swap;
        swap(mplHeap, pMapStruct->mplHeap);
        swap(mplHeapPtr, pMapStruct->mplHeapPtr);
        swap(mplHashTable, pMapStruct->mplHashTable);
        swap(visibleTable, pMapStruct->visibleTable);
        swap(mplVoxelBlocks, pMapStruct->mplVoxelBlocks);
        swap(mplBucketMutex, pMapStruct->mplBucketMutex);
        swap(mpLinkedListHead, pMapStruct->mpLinkedListHead);
        swap(visibleBlockNum, pMapStruct->visibleBlockNum);
    }
}

uint MapStruct::GetNumVisibleBlocks()
{
    uint nVisibleBlock = 0;
    SafeCall(hipMemcpy(&nVisibleBlock, visibleBlockNum, sizeof(uint), hipMemcpyDeviceToHost));
    return nVisibleBlock;
}

void MapStruct::ResetNumVisibleBlocks()
{
    SafeCall(hipMemset(visibleBlockNum, 0, sizeof(uint)));
}

struct FuseMapStruct_functor
{
    HashEntry *plDstEntry;
    HashEntry *plCurrEntry;
    int dstBucketSize;
    int currHashTableSize;
    int dstHashTableSize;
    int *plBucketMutex;
    Voxel *plVoxels;
    Voxel *plDstVoxels;
    int *plHeapPtr;
    int *plHeap;
    int *pLinkedListPtr;

    __device__ __forceinline__ int createNewBlock() const;
    __device__ __forceinline__ void move(Voxel *src, Voxel *dst) const;
    __device__ __forceinline__ void fuse(Voxel *src, Voxel *dst) const;
    __device__ __forceinline__ void operator()() const;
};

__device__ __forceinline__ int FuseMapStruct_functor::createNewBlock() const
{
    int old = atomicSub(plHeapPtr, 1);
    if (old > 0)
    {
        return plHeap[old];
    }
    else
    {
        atomicAdd(plHeapPtr, 1);
        return -1;
    }
}

__device__ __forceinline__ void FuseMapStruct_functor::move(Voxel *src, Voxel *dst) const
{
    memcpy(dst, src, sizeof(Voxel) * BlockSize3);
}

__device__ __forceinline__ void FuseMapStruct_functor::fuse(Voxel *src, Voxel *dst) const
{
    for (int i = 0; i < BlockSize3; ++i)
    {
        float dstSdf = UnPackFloat(dst[i].sdf);
        float srcSdf = UnPackFloat(src[i].sdf);
        dst[i].sdf = PackFloat(dstSdf * dst[i].wt + srcSdf * src[i].wt);
        dst[i].wt = min(255, dst[i].wt + src[i].wt);
    }
}

__device__ __forceinline__ void FuseMapStruct_functor::operator()() const
{
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if (x >= currHashTableSize)
        return;

    if (plCurrEntry[x].ptr == -1)
        return;

    Eigen::Vector3i pos = plCurrEntry[x].pos;
    Voxel *voxels = &plVoxels[plCurrEntry[x].ptr];
    uint hashIdx = hash(pos, dstBucketSize);
    int *mutex = &plBucketMutex[hashIdx];
    HashEntry *current = &plDstEntry[hashIdx];
    HashEntry *empty = nullptr;

    if (current->pos == pos && current->ptr != -1)
    {
        fuse(voxels, &plDstVoxels[current->ptr]);
        return;
    }
    else if (current->ptr == -1)
        empty = current;

    // search through the linked list
    while (current->offset >= 0)
    {
        hashIdx = dstBucketSize + current->offset - 1;
        current = &plDstEntry[hashIdx];
        if (current->pos == pos && current->ptr != -1)
        {
            fuse(voxels, &plDstVoxels[current->ptr]);
            return;
        }
        else if (!empty && current->ptr == -1)
            empty = current;
    }

    // if no existing block is found we create a new one
    if (empty)
    {
        if (LockBucket(mutex))
        {
            int new_ptr = createNewBlock();
            empty->pos = pos;
            empty->ptr = new_ptr;
            move(voxels, &plDstVoxels[new_ptr]);
            UnLockBucket(mutex);
        }
    }
    else
    {
        // we allocate a new one from the linked list
        if (LockBucket(mutex))
        {
            int offset = atomicAdd(pLinkedListPtr, 1);
            if ((offset + dstBucketSize) < dstHashTableSize)
            {
                empty = &plDstEntry[dstBucketSize + offset - 1];
                int new_ptr = createNewBlock();
                empty->ptr = new_ptr;
                empty->pos = pos;
                empty->offset = -1;
                current->offset = offset;
            }
            else
            {
                atomicSub(pLinkedListPtr, 1);
            }

            UnLockBucket(mutex);
        }
    }
}

void MapStruct::Fuse(MapStruct *pMapStruct)
{
    if (!pMapStruct || pMapStruct->empty())
        return;

    if (this->empty())
    {
        this->Swap(pMapStruct);
        return;
    }

    int nHashEntryCom = hashTableSize + pMapStruct->hashTableSize;
    int nBucektCom = static_cast<int>(0.8 * nHashEntryCom);
    int nVoxelBlockCom = voxelBlockSize + pMapStruct->voxelBlockSize;

    MapStruct *pNewMS = new MapStruct(mK);
    pNewMS->create(nHashEntryCom, nBucektCom,
                   nVoxelBlockCom, voxelSize,
                   truncationDist);

    FuseMapStruct_functor functor;
    functor.plDstEntry = pNewMS->mplHashTable;
    functor.plCurrEntry = mplHashTable;
    functor.dstBucketSize = pNewMS->bucketSize;
    functor.currHashTableSize = hashTableSize;
    functor.dstHashTableSize = pNewMS->hashTableSize;
    functor.plBucketMutex = pNewMS->mplBucketMutex;
    functor.plVoxels = mplVoxelBlocks;
    functor.plDstVoxels = pNewMS->mplVoxelBlocks;
    functor.plHeapPtr = pNewMS->mplHeapPtr;
    functor.plHeap = pNewMS->mplHeap;
    functor.pLinkedListPtr = pNewMS->mpLinkedListHead;

    dim3 block(1024);
    dim3 grid(cv::divUp(hashTableSize, block.x));
    callDeviceFunctor<<<grid, block>>>(functor);

    functor.plDstEntry = pNewMS->mplHashTable;
    functor.plCurrEntry = pMapStruct->mplHashTable;
    functor.dstBucketSize = pNewMS->bucketSize;
    functor.currHashTableSize = pMapStruct->hashTableSize;
    functor.dstHashTableSize = pNewMS->hashTableSize;
    functor.plBucketMutex = pNewMS->mplBucketMutex;
    functor.plVoxels = pMapStruct->mplVoxelBlocks;
    functor.plDstVoxels = pNewMS->mplVoxelBlocks;
    functor.plHeapPtr = pNewMS->mplHeapPtr;
    functor.plHeap = pNewMS->mplHeap;
    functor.pLinkedListPtr = pNewMS->mpLinkedListHead;

    block = dim3(1024);
    grid = dim3(cv::divUp(pMapStruct->hashTableSize, block.x));
    callDeviceFunctor<<<grid, block>>>(functor);

    this->Release();
    pMapStruct->Release();
    Swap(pNewMS);
}

struct CreateBlockLineTracingFunctor
{
    int *mplHeap;
    int *mplHeapPtr;
    HashEntry *mplHashTable;
    int *mplBucketMutex;
    int *mpLinkedListHead;
    int hashTableSize;
    int bucketSize;

    float voxelSize;
    float truncDistHalf;
    cv::cuda::PtrStepSz<float> depth;

    float invfx, invfy, cx, cy;
    float depthMin, depthMax;

    Sophus::SE3f T;

    __device__ __forceinline__ void allocateBlock(const Eigen::Vector3i &blockPos) const
    {
        CreateNewBlock(blockPos, mplHeap, mplHeapPtr, mplHashTable, mplBucketMutex, mpLinkedListHead, hashTableSize, bucketSize);
    }

    __device__ __forceinline__ void operator()() const
    {
        const int x = threadIdx.x + blockDim.x * blockIdx.x;
        const int y = threadIdx.y + blockDim.y * blockIdx.y;
        if (x >= depth.cols || y >= depth.rows)
            return;

        float dist = depth.ptr(y)[x];
        if (isnan(dist) || dist < depthMin || dist > depthMax)
            return;

        float distNear = max(depthMin, dist - truncDistHalf);
        float distFar = min(depthMax, dist + truncDistHalf);
        if (distNear >= distFar)
            return;

        Eigen::Vector3i blockStart = VoxelPosToBlockPos(WorldPtToVoxelPos(UnProjectWorld(x, y, distNear, invfx, invfy, cx, cy, T), voxelSize));
        Eigen::Vector3i blockEnd = VoxelPosToBlockPos(WorldPtToVoxelPos(UnProjectWorld(x, y, distFar, invfx, invfy, cx, cy, T), voxelSize));

        Eigen::Vector3i dir = blockEnd - blockStart;
        Eigen::Vector3i increment = Eigen::Vector3i(dir(0) < 0 ? -1 : 1, dir(1) < 0 ? -1 : 1, dir(2) < 0 ? -1 : 1);
        Eigen::Vector3i absIncrement = Eigen::Vector3i(abs(dir(0)), abs(dir(1)), abs(dir(2)));
        Eigen::Vector3i incrementErr = Eigen::Vector3i(absIncrement(0) << 1, absIncrement(1) << 1, absIncrement(2) << 1);

        int err1;
        int err2;

        // Bresenham's line algorithm
        // details see : https://en.m.wikipedia.org/wiki/Bresenham%27s_line_algorithm
        if ((absIncrement(0) >= absIncrement(1)) && (absIncrement(0) >= absIncrement(2)))
        {
            err1 = incrementErr(1) - 1;
            err2 = incrementErr(2) - 1;
            allocateBlock(blockStart);
            for (int i = 0; i < absIncrement(0); ++i)
            {
                if (err1 > 0)
                {
                    blockStart(1) += increment(1);
                    err1 -= incrementErr(0);
                }

                if (err2 > 0)
                {
                    blockStart(2) += increment(2);
                    err2 -= incrementErr(0);
                }

                err1 += incrementErr(1);
                err2 += incrementErr(2);
                blockStart(0) += increment(0);
                allocateBlock(blockStart);
            }
        }
        else if ((absIncrement(1) >= absIncrement(0)) && (absIncrement(1) >= absIncrement(2)))
        {
            err1 = incrementErr(0) - 1;
            err2 = incrementErr(2) - 1;
            allocateBlock(blockStart);
            for (int i = 0; i < absIncrement(1); ++i)
            {
                if (err1 > 0)
                {
                    blockStart(0) += increment(0);
                    err1 -= incrementErr(1);
                }

                if (err2 > 0)
                {
                    blockStart(2) += increment(2);
                    err2 -= incrementErr(1);
                }

                err1 += incrementErr(0);
                err2 += incrementErr(2);
                blockStart(1) += increment(1);
                allocateBlock(blockStart);
            }
        }
        else
        {
            err1 = incrementErr(1) - 1;
            err2 = incrementErr(0) - 1;
            allocateBlock(blockStart);
            for (int i = 0; i < absIncrement(2); ++i)
            {
                if (err1 > 0)
                {
                    blockStart(1) += increment(1);
                    err1 -= incrementErr(2);
                }

                if (err2 > 0)
                {
                    blockStart(0) += increment(0);
                    err2 -= incrementErr(2);
                }

                err1 += incrementErr(1);
                err2 += incrementErr(0);
                blockStart(2) += increment(2);
                allocateBlock(blockStart);
            }
        }
    }
};

struct CheckEntryVisibilityFunctor
{
    HashEntry *mplHashTable;
    HashEntry *visibleEntry;
    uint *visibleEntryCount;
    Sophus::SE3f Tinv;

    int *mplHeap;
    int *mplHeapPtr;
    Voxel *mplVoxelBlocks;
    int cols, rows;
    float fx, fy;
    float cx, cy;
    float depthMin;
    float depthMax;
    float voxelSize;
    int hashTableSize;
    int voxelBlockSize;

    __device__ __forceinline__ void operator()() const
    {
        int idx = threadIdx.x + blockDim.x * blockIdx.x;

        __shared__ bool needScan;

        if (threadIdx.x == 0)
            needScan = false;

        __syncthreads();

        uint increment = 0;
        if (idx < hashTableSize)
        {
            HashEntry *current = &mplHashTable[idx];
            if (current->ptr >= 0)
            {
                bool rval = CheckBlockVisibility(
                    current->pos,
                    Tinv,
                    voxelSize,
                    cols, rows,
                    fx, fy,
                    cx, cy,
                    depthMin,
                    depthMax);

                if (rval)
                {
                    needScan = true;
                    increment = 1;
                }
            }
        }

        __syncthreads();

        if (needScan)
        {
            auto offset = ParallelScan<1024>(increment, visibleEntryCount);
            if (offset >= 0 && offset < hashTableSize && idx < hashTableSize)
                visibleEntry[offset] = mplHashTable[idx];
        }
    }
};

struct DepthFusionFunctor
{

    Voxel *listBlock;
    HashEntry *visible_blocks;

    Sophus::SE3f Tinv;
    float fx, fy;
    float cx, cy;
    float depthMin;
    float depthMax;

    float truncationDist;
    int hashTableSize;
    float voxelSize;
    uint count_visible_block;

    cv::cuda::PtrStepSz<float> depth;

    __device__ __forceinline__ void operator()() const
    {
        if (blockIdx.x >= hashTableSize || blockIdx.x >= count_visible_block)
            return;

        HashEntry &current = visible_blocks[blockIdx.x];
        if (current.ptr == -1)
            return;

        Eigen::Vector3i voxelPos = BlockPosToVoxelPos(current.pos);

#pragma unroll
        for (int blockIdxZ = 0; blockIdxZ < 8; ++blockIdxZ)
        {
            Eigen::Vector3i localPos = Eigen::Vector3i(threadIdx.x, threadIdx.y, blockIdxZ);
            Eigen::Vector3f pt = Tinv * VoxelPosToWorldPt(voxelPos + localPos, voxelSize);

            int u = __float2int_rd(fx * pt(0) / pt(2) + cx + 0.5);
            int v = __float2int_rd(fy * pt(1) / pt(2) + cy + 0.5);
            if (u < 0 || v < 0 || u > depth.cols - 1 || v > depth.rows - 1)
                continue;

            float dist = depth.ptr(v)[u];
            if (isnan(dist) || dist > depthMax || dist < depthMin)
                continue;

            float sdf = dist - pt(2);
            if (sdf < -truncationDist)
                continue;

            sdf = fmin(1.0f, sdf / truncationDist);
            const int localIdx = LocalPosToLocalIdx(localPos);
            Voxel &voxel = listBlock[current.ptr + localIdx];

            auto oldSDF = UnPackFloat(voxel.sdf);
            auto oldWT = voxel.wt;

            if (oldWT == 0)
            {
                voxel.sdf = PackFloat(sdf);
                voxel.wt = 1;
                continue;
            }

            voxel.sdf = PackFloat((oldSDF * oldWT + sdf * 1) / (oldWT + 1));
            voxel.wt = min(255, oldWT + 1);
        }
    }
};

uint MapStruct::CheckNumVisibleBlocks(int cols, int rows, const Sophus::SE3d &Tcm)
{
    ResetNumVisibleBlocks();

    float fx = mK(0, 0);
    float fy = mK(1, 1);
    float cx = mK(0, 2);
    float cy = mK(1, 2);

    CheckEntryVisibilityFunctor functor;
    functor.mplHashTable = mplHashTable;
    functor.mplVoxelBlocks = mplVoxelBlocks;
    functor.visibleEntry = visibleTable;
    functor.visibleEntryCount = visibleBlockNum;
    functor.mplHeap = mplHeap;
    functor.mplHeapPtr = mplHeapPtr;
    functor.voxelBlockSize = voxelBlockSize;
    functor.Tinv = Tcm.inverse().cast<float>();
    functor.cols = cols;
    functor.rows = rows;
    functor.fx = fx;
    functor.fy = fy;
    functor.cx = cx;
    functor.cy = cy;
    functor.depthMin = 0.1f;
    functor.depthMax = 3.0f;
    functor.voxelSize = voxelSize;
    functor.hashTableSize = hashTableSize;

    dim3 block = dim3(1024);
    dim3 grid = dim3(cv::divUp(hashTableSize, block.x));

    callDeviceFunctor<<<grid, block>>>(functor);

    return GetNumVisibleBlocks();
}

void MapStruct::Fuse(cv::cuda::GpuMat depth, const Sophus::SE3d &Tcm)
{
    float fx = mK(0, 0);
    float fy = mK(1, 1);
    float cx = mK(0, 2);
    float cy = mK(1, 2);
    float invfx = 1.0 / mK(0, 0);
    float invfy = 1.0 / mK(1, 1);

    const int cols = depth.cols;
    const int rows = depth.rows;

    CreateBlockLineTracingFunctor step1;
    step1.mplHeap = mplHeap;
    step1.mplHeapPtr = mplHeapPtr;
    step1.mplHashTable = mplHashTable;
    step1.mplBucketMutex = mplBucketMutex;
    step1.mpLinkedListHead = mpLinkedListHead;
    step1.hashTableSize = hashTableSize;
    step1.bucketSize = bucketSize;
    step1.voxelSize = voxelSize;
    step1.truncDistHalf = truncationDist * 0.5;
    step1.depth = depth;
    step1.invfx = invfx;
    step1.invfy = invfy;
    step1.cx = cx;
    step1.cy = cy;
    step1.depthMin = 0.1f;
    step1.depthMax = 3.0f;
    step1.T = Tcm.cast<float>();

    dim3 block(8, 8);
    dim3 grid(cv::divUp(cols, block.x), cv::divUp(rows, block.y));
    callDeviceFunctor<<<grid, block>>>(step1);

    uint nVisibleBlock = CheckNumVisibleBlocks(cols, rows, Tcm);

    if (nVisibleBlock == 0)
        return;

    DepthFusionFunctor step3;
    step3.listBlock = mplVoxelBlocks;
    step3.visible_blocks = visibleTable;
    step3.Tinv = Tcm.inverse().cast<float>();
    step3.fx = fx;
    step3.fy = fy;
    step3.cx = cx;
    step3.cy = cy;
    step3.depthMin = 0.1f;
    step3.depthMax = 3.0f;
    step3.truncationDist = truncationDist;
    step3.hashTableSize = hashTableSize;
    step3.voxelSize = voxelSize;
    step3.count_visible_block = nVisibleBlock;
    step3.depth = depth;

    block = dim3(8, 8);
    grid = dim3(nVisibleBlock);
    callDeviceFunctor<<<grid, block>>>(step3);
}

void MapStruct::SaveToFile(std::string &strFileName)
{
}

void MapStruct::ReadFromFile(std::string &strFileName)
{
}

void MapStruct::Hibernate()
{
    if (mbInHibernation || empty())
        return;

    mpLinkedListHeadHib = new int[1];
    mplHeapPtrHib = new int[1];
    mplBucketMutexHib = new int[bucketSize];
    mplHeapHib = new int[voxelBlockSize];
    mplHashTableHib = new HashEntry[hashTableSize];
    mplVoxelBlocksHib = new Voxel[voxelBlockSize * BlockSize3];

    SafeCall(hipMemcpy(mpLinkedListHeadHib, mpLinkedListHead, sizeof(int), hipMemcpyDeviceToHost));
    SafeCall(hipMemcpy(mplHeapPtrHib, mplHeapPtr, sizeof(int), hipMemcpyDeviceToHost));
    SafeCall(hipMemcpy(mplBucketMutexHib, mplBucketMutex, sizeof(int) * bucketSize, hipMemcpyDeviceToHost));
    SafeCall(hipMemcpy(mplHeapHib, mplHeap, sizeof(int) * voxelBlockSize, hipMemcpyDeviceToHost));
    SafeCall(hipMemcpy(mplHashTableHib, mplHashTable, sizeof(HashEntry) * hashTableSize, hipMemcpyDeviceToHost));
    SafeCall(hipMemcpy(mplVoxelBlocksHib, mplVoxelBlocks, sizeof(Voxel) * voxelBlockSize * BlockSize3, hipMemcpyDeviceToHost));

    mbInHibernation = true;

    SafeCall(hipFree((void *)mplHeap));
    SafeCall(hipFree((void *)mplHeapPtr));
    SafeCall(hipFree((void *)mplHashTable));
    SafeCall(hipFree((void *)mplBucketMutex));
    SafeCall(hipFree((void *)mpLinkedListHead));
    SafeCall(hipFree((void *)mplVoxelBlocks));
    SafeCall(hipFree((void *)visibleBlockNum));
    SafeCall(hipFree((void *)visibleTable));
}

void MapStruct::ReActivate()
{
    if (!mbInHibernation || empty())
        return;

    SafeCall(hipMalloc((void **)&mpLinkedListHead, sizeof(int)));
    SafeCall(hipMalloc((void **)&mplHeapPtr, sizeof(int)));
    SafeCall(hipMalloc((void **)&visibleBlockNum, sizeof(uint)));
    SafeCall(hipMalloc((void **)&mplBucketMutex, sizeof(int) * bucketSize));
    SafeCall(hipMalloc((void **)&mplHeap, sizeof(int) * voxelBlockSize));
    SafeCall(hipMalloc((void **)&mplHashTable, sizeof(HashEntry) * hashTableSize));
    SafeCall(hipMalloc((void **)&visibleTable, sizeof(HashEntry) * hashTableSize));
    SafeCall(hipMalloc((void **)&mplVoxelBlocks, sizeof(Voxel) * voxelBlockSize * BlockSize3));

    SafeCall(hipMemcpy(mpLinkedListHead, mpLinkedListHeadHib, sizeof(int), hipMemcpyHostToDevice));
    SafeCall(hipMemcpy(mplHeapPtr, mplHeapPtrHib, sizeof(int), hipMemcpyHostToDevice));
    SafeCall(hipMemcpy(mplBucketMutex, mplBucketMutexHib, sizeof(int) * bucketSize, hipMemcpyHostToDevice));
    SafeCall(hipMemcpy(mplHeap, mplHeapHib, sizeof(int) * voxelBlockSize, hipMemcpyHostToDevice));
    SafeCall(hipMemcpy(mplHashTable, mplHashTableHib, sizeof(HashEntry) * hashTableSize, hipMemcpyHostToDevice));
    SafeCall(hipMemcpy(mplVoxelBlocks, mplVoxelBlocksHib, sizeof(Voxel) * voxelBlockSize * BlockSize3, hipMemcpyHostToDevice));

    mbInHibernation = false;

    delete mpLinkedListHeadHib;
    delete mplHeapPtrHib;
    delete mplBucketMutexHib;
    delete mplHeapHib;
    delete mplHashTableHib;
    delete mplVoxelBlocksHib;
}

uint MapStruct::GetVisibleBlocks()
{
    uint temp = 0;
    SafeCall(hipMemcpy(&temp, visibleBlockNum, sizeof(uint), hipMemcpyDeviceToHost));
    return temp;
}

void MapStruct::ResetVisibleBlocks()
{
    SafeCall(hipMemset(visibleBlockNum, 0, sizeof(uint)));
}

void MapStruct::RayTrace(const Sophus::SE3d &Tcm)
{
    if (mpRayTraceEngine)
    {
        mpRayTraceEngine->RayTrace(this, Tcm);
    }
}

cv::cuda::GpuMat MapStruct::GetRayTracingResult()
{
    return mpRayTraceEngine->GetVMap();
}

void MapStruct::SetActiveFlag(bool flag)
{
    std::unique_lock<std::mutex> lock(mMutexActive);
    mbActive = flag;
}

bool MapStruct::isActive()
{
    std::unique_lock<std::mutex> lock(mMutexActive);
    return mbActive;
}